#include "hip/hip_runtime.h"
/* batched_gpu_to_cpu_bench.cu
 * make
 * CUDA_MODULE_LOADING=EAGER ./batched_gpu_to_cpu_bench
 */

#include <atomic>
#include <chrono>
#include <thread>
#include <tuple>
#include <vector>
#include <assert.h>
#if defined(__x86_64__) || defined(__i386__)
#include <immintrin.h>
#endif
#include "common.hpp"
#include <stdint.h>
#include <stdio.h>
#include <unistd.h>

struct alignas(128) Fifo {
  // Using volatile and avoiding atomics.
  uint64_t head;                      // Next slot to produce
  uint64_t tail;                      // Next slot to consume
  uint64_t volatile buf[kQueueSize];  // Payload buffer (8 bytes).
};

__device__ __forceinline__ uint64_t ld_volatile(uint64_t* ptr) {
#if defined(__CUDA_ARCH__) || defined(__HIP_DEVICE_COMPILE__)
#ifdef __CUDA_ARCH__
  uint64_t ans;
  asm volatile("ld.volatile.global.u64 %0, [%1];"
               : "=l"(ans)
               : "l"(ptr)
               : "memory");
  return ans;
#elif defined(__HIP_DEVICE_COMPILE__)
  uint64_t ans;
  ans = __builtin_nontemporal_load(ptr);
  return ans;
#else
#error "Not supported"
#endif
#else
  return *((volatile uint64_t const*)ptr);
#endif
}

__device__ unsigned long long cycle_accum[kNumThBlocks] = {0};
__device__ unsigned int op_count[kNumThBlocks] = {0};

__global__ void gpu_issue_batched_commands(Fifo* fifos) {
  int const bid = blockIdx.x;
  int const tid = threadIdx.x;
  Fifo* my_fifo = &fifos[bid];
  if (tid != 0) {
    return;
  }

  uint32_t complete = 0;
  __shared__ unsigned long long cycle_accum_smem;
  __shared__ unsigned int op_count_smem;
  cycle_accum_smem = 0ull;
  op_count_smem = 0u;

  extern __shared__ unsigned long long start_cycle_smem[];

  for (int it = 0; it < kIterations; it += kBatchSize) {
    uint64_t my_hdr;
    uint64_t cur_tail;

    unsigned int const todo =
        (it + kBatchSize <= kIterations) ? kBatchSize : (kIterations - it);

    while (true) {
      // CPU does not modify the head.
      uint64_t cur_head = my_fifo->head;
      cur_tail = ld_volatile(&my_fifo->tail);
      if (cur_head - cur_tail + todo <= kQueueSize) {
        my_fifo->head = cur_head + todo;
        my_hdr = cur_head;
        break;
        // if (atomicAdd_system(&(my_fifo->head), todo) == cur_head) {
        // my_hdr = cur_head;
        // break; // Successfully reserved a slot
        // }
      }
    }

#pragma unroll
    for (int i = 0; i < todo; ++i) {
      unsigned long long idx = (my_hdr + i) & kQueueMask;
      unsigned long long t0 = clock64();
      unsigned long long cmd =
          (static_cast<uint64_t>(bid) << 32) | (it + i + 1);

      start_cycle_smem[idx] = t0;
      my_fifo->buf[idx] = cmd;
    }
    __threadfence_system();
    while (complete < my_hdr + todo) {
      uint32_t cidx = complete & kQueueMask;
      if (complete < my_fifo->tail) {
        unsigned long long t1 = clock64();
        unsigned long long cycles = t1 - start_cycle_smem[cidx];
        cycle_accum_smem += cycles;
        op_count_smem++;
        complete++;
      } else {
        break;
      }
    }
  }
  while (complete < kIterations) {
    uint32_t cidx = complete & kQueueMask;
    while (complete >= my_fifo->tail) { /* spin */
    }

    unsigned long long t1 = clock64();
    cycle_accum_smem += (t1 - start_cycle_smem[cidx]);
    ++op_count_smem;
    ++complete;
  }

  cycle_accum[bid] = cycle_accum_smem;
  op_count[bid] = op_count_smem;
}

void cpu_proxy(Fifo* fifo, int thread_idx) {
  // printf("CPU thread for block %d started\n", thread_idx);
  pin_thread_to_cpu(thread_idx);

  uint64_t my_tail = 0;
  for (int seen = 0; seen < kIterations; ++seen) {
    // TODO: here, if CPU caches fifo->head, it may not see the updates from
    // GPU.
    while (fifo->head == my_tail) {
      cpu_relax();
    }
    uint64_t idx = my_tail & kQueueMask;
    uint64_t cmd;
    do {
      cmd = fifo->buf[idx];
      cpu_relax();  // Avoid hammering the cacheline.
    } while (cmd == 0);
    uint64_t expected_cmd =
        (static_cast<uint64_t>(thread_idx) << 32) | (seen + 1);
    if (cmd != expected_cmd) {
      fprintf(stderr, "Error: block %d, expected cmd %llu, got %llu\n",
              thread_idx, static_cast<unsigned long long>(expected_cmd),
              static_cast<unsigned long long>(cmd));
      exit(1);
    }
    fifo->buf[idx] = 0;
    // std::atomic_thread_fence(std::memory_order_release);
    my_tail++;
    fifo->tail = my_tail;
    // _mm_clflush(&(fifo->tail));
  }
}

int main() {
  gpuStream_t stream1;
  GPU_RT_CHECK(gpuStreamCreate(&stream1));

  gpuDeviceProp prop;
  gpuGetDeviceProperties(&prop, 0);
  printf("clock rate: %d kHz\n", prop.clockRate);

  Fifo* fifos;
  GPU_RT_CHECK(
      gpuHostAlloc(&fifos, sizeof(Fifo) * kNumThBlocks, gpuHostAllocMapped));

  for (int i = 0; i < kNumThBlocks; ++i) {
    fifos[i].head = 0;
    fifos[i].tail = 0;
    for (uint32_t j = 0; j < kQueueSize; ++j) {
      fifos[i].buf[j] = 0;  // Initialize the buffer
    }
  }

  // Launch one CPU polling thread per block
  std::vector<std::thread> cpu_threads;
  for (int i = 0; i < kNumThBlocks; ++i) {
    cpu_threads.emplace_back(cpu_proxy, &fifos[i], i);
  }
  auto t0 = std::chrono::high_resolution_clock::now();
  size_t shmem_bytes = kQueueSize * sizeof(unsigned long long);
  gpu_issue_batched_commands<<<kNumThBlocks, kNumThPerBlock, shmem_bytes,
                               stream1>>>(fifos);
  GPU_RT_CHECK_ERRORS("gpu_issue_command kernel failed");
  GPU_RT_CHECK(gpuStreamSynchronize(stream1));
  auto t1 = std::chrono::high_resolution_clock::now();

  for (auto& t : cpu_threads) {
    t.join();
  }

  unsigned long long h_cycles[kNumThBlocks];
  unsigned int h_ops[kNumThBlocks];
  gpuMemcpyFromSymbol(h_cycles, cycle_accum, sizeof(h_cycles));
  gpuMemcpyFromSymbol(h_ops, op_count, sizeof(h_ops));

  unsigned int tot_ops = 0;
  double total_us = 0;
  unsigned long long tot_cycles = 0;
  printf("\nPer-block avg latency:\n");
  for (int b = 0; b < kNumThBlocks; ++b) {
    double us = (double)h_cycles[b] * 1000.0 / prop.clockRate / h_ops[b];
    printf("  Block %d : %.3f µs over %u ops\n", b, us, h_ops[b]);
    total_us += us;
    tot_cycles += h_cycles[b];
    tot_ops += h_ops[b];
  }
  double wall_ms = std::chrono::duration<double, std::milli>(t1 - t0).count();
  double throughput = (double)(kNumThBlocks * kIterations) / (wall_ms * 1000.0);

  printf("\nOverall avg GPU-measured latency  : %.3f µs\n",
         (double)tot_cycles * 1000.0 / prop.clockRate / tot_ops);
  printf("Total cycles                       : %llu\n", tot_cycles);
  printf("Total ops                          : %u\n", tot_ops);
  printf("End-to-end Wall-clock time        : %.3f ms\n", wall_ms);
  printf("Throughput                        : %.2f Mops/s\n", throughput);

  GPU_RT_CHECK(gpuFreeHost(fifos));
  GPU_RT_CHECK(gpuStreamDestroy(stream1));
}