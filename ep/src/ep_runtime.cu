#include "hip/hip_runtime.h"
#include "ep_launch.cuh"
#include "ep_runtime.cuh"
#include "ep_utils.cuh"
#include <iostream>

namespace internode {

int init(std::vector<uint8_t> const& root_unique_id_val, int rank,
         int num_ranks, bool low_latency_mode) {
  std::cout << "[internode::init] dummy init invoked" << std::endl;
  return 0;
}

void* alloc(std::size_t size, std::size_t alignment) {
  // NOTE(MaoZiming): alignment is ignored here since hipMalloc already aligns
  // to at least 256 bytes
  void* ptr = nullptr;
  hipError_t err = hipMalloc(&ptr, size);
  if (err != hipSuccess) {
    std::cerr << "[internode::alloc] hipMalloc failed: "
              << hipGetErrorString(err) << std::endl;
    return nullptr;
  }
  std::cout << "[internode::alloc] allocated " << size << " bytes at " << ptr
            << std::endl;
  return ptr;
}

void finalize() {
  std::cout << "[internode::finalize] dummy finalize invoked" << std::endl;
}

void barrier() {
  std::cout << "[internode::barrier] dummy barrier invoked" << std::endl;
}

void free(void* ptr) {
  std::cout << "[internode::free] dummy free invoked" << std::endl;
  // free
  hipError_t err = hipFree(ptr);
  if (err != hipSuccess) {
    std::cerr << "[internode::free] hipFree failed: "
              << hipGetErrorString(err) << std::endl;
  } else {
    std::cout << "[internode::free] freed memory at " << ptr << std::endl;
  }
}

std::vector<uint8_t> get_unique_id() { return std::vector<uint8_t>(64, 0); }

}  // namespace internode

namespace intranode {

template <int kNumRanks>
__global__ void barrier(int** barrier_signal_ptrs, int rank) {
  barrier_block<kNumRanks>(barrier_signal_ptrs, rank);
}

void barrier(int** barrier_signal_ptrs, int rank, int num_ranks,
             hipStream_t stream) {
#define BARRIER_LAUNCH_CASE(ranks)                                \
  LAUNCH_KERNEL(&cfg, barrier<ranks>, barrier_signal_ptrs, rank); \
  break

  SETUP_LAUNCH_CONFIG(1, 32, stream);
  SWITCH_RANKS(BARRIER_LAUNCH_CASE);
#undef BARRIER_LAUNCH_CASE
}

}  // namespace intranode