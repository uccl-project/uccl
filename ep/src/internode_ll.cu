#include "hip/hip_runtime.h"
#include "ep_configs.cuh"
#include "ep_launch.cuh"
#include "ep_runtime.cuh"
#include "ep_util.hpp"
#include "ep_utils.cuh"
#include "internode_ll.cuh"
#include "uccl_ibgda.cuh"
#include <iostream>
#include <vector>
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;
namespace uccl {
namespace internode_ll {

template <int kNumThreads>
__launch_bounds__(kNumThreads, 1) __global__
    void clean_low_latency_buffer(int* clean_0, int num_clean_int_0,
                                  int* clean_1, int num_clean_int_1) {
  // Barrier before cleaning (in case of unfinished chunked EP)
  // nvshmemx_barrier_all_block();

  // Clean
  auto thread_id = static_cast<int>(threadIdx.x);
#pragma unroll
  for (int i = thread_id; i < num_clean_int_0; i += kNumThreads) clean_0[i] = 0;
#pragma unroll
  for (int i = thread_id; i < num_clean_int_1; i += kNumThreads) clean_1[i] = 0;

  // Barrier after cleaning (make sure the low-latency mode works fine)
  // nvshmemx_barrier_all_block();
}

void clean_low_latency_buffer(int* clean_0, int num_clean_int_0, int* clean_1,
                              int num_clean_int_1, hipStream_t stream) {
  constexpr int kNumThreads = 256;

  SETUP_LAUNCH_CONFIG(1, kNumThreads, stream);
  LAUNCH_KERNEL(&cfg, clean_low_latency_buffer<kNumThreads>, clean_0,
                num_clean_int_0, clean_1, num_clean_int_1);
}

template <bool kUseFP8, bool kUseUE8M0, int kHidden>
__global__ __launch_bounds__(1024, 1) void dispatch(
    void* packed_recv_x, void* packed_recv_x_scales, int* packed_recv_src_info,
    int64_t* packed_recv_layout_range, int* packed_recv_count,
    int* cumulative_local_expert_recv_stats,
    int64_t* dispatch_wait_recv_cost_stats, void* rdma_recv_x,
    int* rdma_recv_count, void* rdma_x, void const* x, int64_t const* topk_idx,
    int* atomic_counter_per_expert, int* atomic_finish_counter_per_expert,
    int* next_clean, int* next_clean_second, int num_next_clean_int,
    int num_tokens, int num_max_dispatch_tokens_per_rank, int num_topk,
    int num_experts, int rank, int num_ranks, int num_warp_groups,
    int num_warps_per_group, bool round_scale, int phases,
    uint64_t const* ring_addrs, int num_ring_addrs, int max_nvl_peers,
    int low_latency_buffer_idx, void** ipc_rdma_base_ptrs = nullptr,
    void* rdma_buffer_ptr = nullptr, void* atomic_buffer_ptr = nullptr,
    int* rdma_recv_count_internode = nullptr) {
  auto const sm_id = static_cast<int>(blockIdx.x);
  auto const thread_id = static_cast<int>(threadIdx.x);
  auto const warp_id = thread_id / 32, lane_id = get_lane_id();
  auto const num_sms = static_cast<int>(gridDim.x);
  auto const num_warps = num_warp_groups * num_warps_per_group;
  auto const num_local_experts = num_experts / num_ranks;
  auto const warp_group_id = warp_id / num_warps_per_group;
  auto const sub_warp_id = warp_id % num_warps_per_group;
  auto const responsible_expert_idx = sm_id * num_warp_groups + warp_group_id;

  // May extract UE8M0 from the scales
  using scale_t = std::conditional_t<kUseUE8M0, uint8_t, float>;
  using packed_t = std::conditional_t<kUseUE8M0, uint32_t, float>;
  EP_STATIC_ASSERT(sizeof(packed_t) % sizeof(scale_t) == 0,
                   "Invalid vector length");

  // FP8 staffs
  constexpr int kNumPerChannels = 128;
  int const num_scales = kHidden / kNumPerChannels;
  size_t const hidden_bytes =
      kHidden * (kUseFP8 ? sizeof(__hip_fp8_storage_t) : sizeof(hip_bfloat16));
  size_t const hidden_int4 = hidden_bytes / sizeof(int4);

  // Message package: hidden data, FP8 scales, index at source
  // NOTES: currently we have 3 reserved int fields for future use
  using vec_t = typename std::conditional<kUseFP8, int2, int4>::type;
  size_t const num_bytes_per_msg =
      sizeof(int4) + (kUseFP8 ? (kHidden + num_scales * sizeof(float))
                              : (kHidden * sizeof(hip_bfloat16)));
  size_t const num_int4_per_msg = num_bytes_per_msg / sizeof(int4);
  EP_DEVICE_ASSERT(num_bytes_per_msg % sizeof(int4) == 0);

  // Expert counts
  constexpr int kNumMaxWarpGroups = 32;
  __shared__ int shared_num_tokens_sent_per_expert[kNumMaxWarpGroups];

  // Sending phase
  if ((phases & LOW_LATENCY_SEND_PHASE) == 0) goto LOW_LATENCY_DISPATCH_RECV;

  // There are 2 kinds of warps in this part:
  // 1. The first-kind warps for FP8 cast and sending top-k tokens
  // 2. The last warp for reading `topk_idx` and count for per-expert
  // information
  if (warp_id < num_warps - 1) {
    constexpr int kNumElemsPerRead = sizeof(int4) / sizeof(hip_bfloat16);
    EP_STATIC_ASSERT(kHidden % (32 * kNumElemsPerRead) == 0, "Invalid hidden");
    EP_STATIC_ASSERT(kNumElemsPerRead * 32 % kNumPerChannels == 0,
                     "Invalid vectorization");
    auto const num_threads = (num_warps - 1) * 32;
    size_t const hidden_bf16_int4 = kHidden / kNumElemsPerRead;

    for (int token_idx = sm_id; token_idx < num_tokens; token_idx += num_sms) {
      auto const x_int4 =
          static_cast<int4 const*>(x) + token_idx * hidden_bf16_int4;
      auto const rdma_x_src_idx = reinterpret_cast<int*>(
          static_cast<uint8_t*>(rdma_x) + token_idx * num_bytes_per_msg);
      auto const rdma_x_vec = reinterpret_cast<vec_t*>(
          reinterpret_cast<uint8_t*>(rdma_x_src_idx) + sizeof(int4));
      auto const rdma_x_scales = reinterpret_cast<float*>(
          reinterpret_cast<uint8_t*>(rdma_x_vec) + hidden_bytes);

      // Overlap top-k index read and source token index writes
      auto dst_expert_idx =
          warp_id < num_topk ? static_cast<int>(__ldg(
                                   topk_idx + token_idx * num_topk + warp_id))
                             : -1;
      thread_id == 0 ? (*rdma_x_src_idx = token_idx) : 0;

// FP8 cast
#pragma unroll
      for (int i = thread_id; i < hidden_bf16_int4; i += num_threads) {
        // Read
        auto int4_value = __ldg(x_int4 + i);

        if constexpr (kUseFP8) {
          // Calculate local amax
          auto bf16_values = reinterpret_cast<hip_bfloat16*>(&int4_value);
          float fp32_values[kNumElemsPerRead];
          float amax = kFP8Margin, scale, scale_inv;
#pragma unroll
          for (int j = 0; j < kNumElemsPerRead; ++j) {
            fp32_values[j] = static_cast<float>(bf16_values[j]);
            amax = fmaxf(amax, fabsf(fp32_values[j]));
          }

          // Reduce amax and scale
          EP_STATIC_ASSERT(kNumElemsPerRead * 32 / kNumPerChannels == 2,
                           "Invalid vectorization");
          amax = warp_reduce_max<16>(amax);
          calculate_fp8_scales(amax, scale, scale_inv, round_scale);
          if (lane_id == 0 or lane_id == 16)
            rdma_x_scales[i * kNumElemsPerRead / 128] = scale_inv;

          // Cast into send buffer
          vec_t int2_value;
          auto fp8x2_values =
              reinterpret_cast<__hip_fp8x2_storage_t*>(&int2_value);
#pragma unroll
          for (int j = 0; j < kNumElemsPerRead; j += 2) {
            float2 fp32x2 = {fp32_values[j] * scale,
                             fp32_values[j + 1] * scale};
            fp8x2_values[j / 2] =
                __hip_cvt_float2_to_fp8x2(fp32x2, __HIP_SATFINITE, __HIP_E4M3_FNUZ);
          }
          rdma_x_vec[i] = int2_value;
        } else {
          // Reinterpret-cast is for C++14 compatibility
          rdma_x_vec[i] = *reinterpret_cast<vec_t*>(&int4_value);
        }
      }
      sync_barrier_1(num_threads);

      // Issue IBGDA sends
      if (dst_expert_idx >= 0) {
        int slot_idx =
            lane_id == 0
                ? atomicAdd(atomic_counter_per_expert + dst_expert_idx, 1)
                : 0;
        slot_idx = __shfl_sync(0xffffffff, slot_idx, 0);
        auto const dst_rank = dst_expert_idx / num_local_experts;
        auto const dst_expert_local_idx = dst_expert_idx % num_local_experts;
        auto const src_ptr = reinterpret_cast<uint64_t>(rdma_x_src_idx);
        auto const dst_ptr =
            reinterpret_cast<uint64_t>(rdma_recv_x) +
            dst_expert_local_idx * num_ranks *
                num_max_dispatch_tokens_per_rank * num_bytes_per_msg +
            rank * num_max_dispatch_tokens_per_rank * num_bytes_per_msg +
            slot_idx * num_bytes_per_msg;
        auto const dst_p2p_ptr =
            ipc_rdma_base_ptrs
                ? uccl::get_ipc_p2p_ptr(dst_ptr, ipc_rdma_base_ptrs, rank,
                                        dst_rank, max_nvl_peers, 0)
                : 0;
        if (dst_p2p_ptr == 0) {
          __threadfence_system();
          uccl::nvshmemi_ibgda_put_nbi_warp(
              dst_ptr - reinterpret_cast<uint64_t>(rdma_buffer_ptr), src_ptr,
              num_bytes_per_msg, dst_rank,
              /*warp_id=*/dst_expert_local_idx,  // NOTE(Yang): for selecting
                                                 // rb.
              lane_id, slot_idx, ring_addrs, num_ring_addrs, false,
              low_latency_buffer_idx);
        } else {
          // Intra-node: use direct memory copy via IPC
          auto const* src_int4_ptr = reinterpret_cast<int4 const*>(src_ptr);
          auto* dst_int4_ptr = reinterpret_cast<int4*>(dst_p2p_ptr);
          UNROLLED_WARP_COPY(8, lane_id, num_int4_per_msg, dst_int4_ptr,
                             src_int4_ptr, ld_cg_global, st_cg_global);
        }
        // Increase counter after finishing
        __syncwarp();
        lane_id == 0 ? atomic_add_release_global(
                           atomic_finish_counter_per_expert + dst_expert_idx, 1)
                     : 0;
      }
    }
  } else if (warp_id == num_warps - 1) {
    // NOTE(MaoZiming): These checks are ibgda specific.
    EP_DEVICE_ASSERT(num_sms > 1);
    if (sm_id == 0) {
      // The first SM is also responsible for cleaning the next buffer
#pragma unroll
      for (int i = lane_id; i < num_next_clean_int; i += 32) {
        next_clean[i] = 0;
        next_clean_second[i] = 0;
      }
      // Notify before executing `int_p`
      __syncwarp();
#pragma unroll
      for (int i = lane_id; i < num_experts; i += 32)
        atomic_add_release_global(atomic_finish_counter_per_expert + i,
                                  FINISHED_SUM_TAG);
    }
    // This SM should be responsible for some destination experts, read
    // `topk_idx` for them
    int expert_count[kNumMaxWarpGroups] = {0};
    auto const expert_begin_idx = sm_id * num_warp_groups;
    auto const expert_end_idx =
        min(expert_begin_idx + num_warp_groups, num_experts);

// Per lane count
#pragma unroll 8
    for (int i = lane_id; i < num_tokens * num_topk; i += 32) {
      auto idx = static_cast<int>(__ldg(topk_idx + i));
      if (idx >= expert_begin_idx and idx < expert_end_idx)
        expert_count[idx - expert_begin_idx]++;
    }

// Warp reduce
#pragma unroll
    for (int i = expert_begin_idx; i < expert_end_idx; ++i) {
      auto sum = warp_reduce_sum(expert_count[i - expert_begin_idx]);
      if (lane_id == 0) {
        shared_num_tokens_sent_per_expert[i - expert_begin_idx] = sum;
        atomic_add_release_global(atomic_finish_counter_per_expert + i,
                                  FINISHED_SUM_TAG - sum);
      }
    }
  }
  __syncthreads();
  // Issue count sends
  if (responsible_expert_idx < num_experts and sub_warp_id == 0 and
      lane_id == 0) {
    auto const dst_rank = responsible_expert_idx / num_local_experts;
    auto const dst_expert_local_idx =
        responsible_expert_idx % num_local_experts;
    auto const num_tokens_sent =
        shared_num_tokens_sent_per_expert[responsible_expert_idx -
                                          sm_id * num_warp_groups];
    // Wait local sends issued and send expert counts
    while (ld_acquire_global(atomic_finish_counter_per_expert +
                             responsible_expert_idx) != FINISHED_SUM_TAG * 2)
      ;

    auto dst_ptr = reinterpret_cast<uint64_t>(
        rdma_recv_count + dst_expert_local_idx * num_ranks + rank);
    auto dst_ptr_internode = reinterpret_cast<uint64_t>(
        rdma_recv_count_internode + dst_expert_local_idx * num_ranks + rank);
    // Try to use IPC for intra-node atomic operations
    auto const dst_p2p_ptr =
        ipc_rdma_base_ptrs
            ? uccl::get_ipc_p2p_ptr(dst_ptr, ipc_rdma_base_ptrs, rank, dst_rank,
                                    max_nvl_peers, 0)
            : 0;
    if (dst_p2p_ptr == 0) {
      // Inter-node or no IPC: use IBGDA atomic
      uccl::nvshmemi_ibgda_amo_nonfetch_add(
          dst_ptr_internode - reinterpret_cast<uint64_t>(atomic_buffer_ptr),
          -num_tokens_sent - 1, dst_rank,
          /*qp_id=*/-1,                      // NOTE(Yang): not used.
          /*warp_id=*/dst_expert_local_idx,  // NOTE(Yang): for selecting rb.
          false, ring_addrs, num_ring_addrs, false, low_latency_buffer_idx);

    } else {
      // Intra-node: use direct atomic operation
      st_release_sys_global(reinterpret_cast<int*>(dst_p2p_ptr),
                            -num_tokens_sent - 1);
    }
    // Clean workspace for next use
    atomic_counter_per_expert[responsible_expert_idx] = 0;
    atomic_finish_counter_per_expert[responsible_expert_idx] = 0;

    // Clean `packed_recv_count`
    if (dst_rank == 0) packed_recv_count[dst_expert_local_idx] = 0;
  }
  __syncwarp();

// Receiving phase
LOW_LATENCY_DISPATCH_RECV:
  if ((phases & LOW_LATENCY_RECV_PHASE) == 0) {
    return;
  }

  // For send-and-recv kernels, we need a grid sync for making
  // `packed_recv_count` visible
  if (phases & LOW_LATENCY_SEND_PHASE) cg::this_grid().sync();

  // Receiving and packing
  if (responsible_expert_idx < num_experts) {
    auto const src_rank = responsible_expert_idx / num_local_experts;
    auto const local_expert_idx = responsible_expert_idx % num_local_experts;
    auto const rdma_recv_x_uint8 =
        static_cast<uint8_t*>(rdma_recv_x) +
        local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank *
            num_bytes_per_msg +
        src_rank * num_max_dispatch_tokens_per_rank * num_bytes_per_msg;
    auto const recv_x_int4 = static_cast<int4*>(packed_recv_x) +
                             local_expert_idx * num_ranks *
                                 num_max_dispatch_tokens_per_rank * hidden_int4;
    auto const recv_src_info =
        packed_recv_src_info +
        local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank;
    auto const recv_range =
        packed_recv_layout_range + local_expert_idx * num_ranks;
    auto const num_aligned_scales =
        align<int>(num_scales, sizeof(float) / sizeof(scale_t));
    auto const recv_x_scales = static_cast<scale_t*>(packed_recv_x_scales) +
                               local_expert_idx * num_ranks *
                                   num_max_dispatch_tokens_per_rank *
                                   num_aligned_scales;

    // Shared between sub-warps in warp groups
    __shared__ int shared_num_recv_tokens[kNumMaxWarpGroups],
        shared_recv_token_begin_idx[kNumMaxWarpGroups];

    // Wait tokens to arrive
    // NOTES: using sub-warp 1 to overlap with sub-warp 0
    int num_recv_tokens_internode = 0, num_recv_tokens_ipc = 0,
        num_recv_tokens = 0, recv_token_begin_idx = 0;
    EP_DEVICE_ASSERT(num_warps_per_group > 1 and num_warp_groups < 15);
    if (sub_warp_id == 1 and lane_id == 0) {
      auto start_time = clock64();
      while ((src_rank / max_nvl_peers == rank / max_nvl_peers) &&
             (num_recv_tokens_ipc = ld_acquire_sys_global(
                  rdma_recv_count + local_expert_idx * num_ranks + src_rank)) ==
                 0)
        ;
      while ((src_rank / max_nvl_peers != rank / max_nvl_peers) &&
             (num_recv_tokens_internode = ld_acquire_sys_global(
                  rdma_recv_count_internode + local_expert_idx * num_ranks +
                  src_rank)) == 0)
        ;

      if (src_rank / max_nvl_peers == rank / max_nvl_peers) {
        if (ld_acquire_sys_global(rdma_recv_count_internode +
                                  local_expert_idx * num_ranks + src_rank) !=
            0) {
          printf(
              "Same node but rdma_recv_count_internode is not zero! src_rank: "
              "%d, rank: %d, max_nvl_peers: %d\n",
              src_rank, rank, max_nvl_peers);
          assert(false);
        }
      }
      if (src_rank / max_nvl_peers != rank / max_nvl_peers) {
        if (ld_acquire_sys_global(rdma_recv_count +
                                  local_expert_idx * num_ranks + src_rank) !=
            0) {
          printf(
              "Different node but rdma_recv_count is not zero! src_rank: %d, "
              "rank: %d, max_nvl_peers: %d\n",
              src_rank, rank, max_nvl_peers);
          assert(false);
        }
      }
      auto wait_recv_cost = clock64() - start_time;
      num_recv_tokens_internode =
          num_recv_tokens_internode != 0 ? -num_recv_tokens_internode - 1 : 0;
      num_recv_tokens_ipc =
          num_recv_tokens_ipc != 0 ? -num_recv_tokens_ipc - 1 : 0;
      // printf(
      //     "num_recv_tokens_internode: %d, num_recv_tokens_ipc: %d, src_rank:"
      //     "%d, rank: %d, max_nvl_peers: %d, responsible_expert_idx: %d,"
      //     "num_experts: %d, num_local_experts: %d\n",
      //     num_recv_tokens_internode, num_recv_tokens_ipc, src_rank, rank,
      //     max_nvl_peers, responsible_expert_idx, num_experts,
      //     num_local_experts);
      num_recv_tokens = num_recv_tokens_internode + num_recv_tokens_ipc;
      recv_token_begin_idx =
          atomicAdd(packed_recv_count + local_expert_idx, num_recv_tokens);
      shared_num_recv_tokens[warp_group_id] = num_recv_tokens;
      shared_recv_token_begin_idx[warp_group_id] = recv_token_begin_idx;
      recv_range[src_rank] =
          pack2<int, int64_t>(num_recv_tokens, recv_token_begin_idx);
      // Add stats for diagnosis
      if (cumulative_local_expert_recv_stats != nullptr)
        atomicAdd(cumulative_local_expert_recv_stats + local_expert_idx,
                  num_recv_tokens);
      if (dispatch_wait_recv_cost_stats != nullptr)
        atomicAdd(reinterpret_cast<unsigned long long*>(
                      dispatch_wait_recv_cost_stats + src_rank),
                  wait_recv_cost);
    }
    sync_barrier(warp_group_id + 2, num_warps_per_group * 32);
    num_recv_tokens = shared_num_recv_tokens[warp_group_id];
    recv_token_begin_idx = shared_recv_token_begin_idx[warp_group_id];

    // Copy tokens
    EP_DEVICE_ASSERT(num_scales <= 64);
    for (int i = sub_warp_id; i < num_recv_tokens; i += num_warps_per_group) {
      // Copy source info
      auto const src_src_idx =
          reinterpret_cast<int*>(rdma_recv_x_uint8 + i * num_bytes_per_msg);
      if (lane_id == 0)
        recv_src_info[recv_token_begin_idx + i] = ld_cg_global(src_src_idx);
      __syncwarp();

      // Copy data
      // NOTES: only 2 load iterations for 7K hidden with 7 unrolls
      auto const src_data = reinterpret_cast<int4*>(
          reinterpret_cast<uint8_t*>(src_src_idx) + sizeof(int4));
      auto const dst_data =
          recv_x_int4 + (recv_token_begin_idx + i) * hidden_int4;
      UNROLLED_WARP_COPY(7, lane_id, hidden_int4, dst_data, src_data,
                         ld_cg_global, st_cg_global);

      // Copy scales
      if constexpr (kUseFP8) {
        // Equivalent CuTe layout:
        //   (num_tokens, (num_packed, num_elems_per_pack)):(num_elems_per_pack,
        //   (num_tokens * num_elems_per_pack, 1))
        auto const src_scales = reinterpret_cast<float*>(
            reinterpret_cast<uint8_t*>(src_data) + hidden_bytes);
        auto const num_elems_per_pack =
            static_cast<int>(sizeof(packed_t) / sizeof(scale_t));
        auto const token_idx = recv_token_begin_idx + i;
        auto const token_stride = num_elems_per_pack;
        auto const pack_stride =
            num_ranks * num_max_dispatch_tokens_per_rank * num_elems_per_pack;
        if (lane_id < num_scales) {
          auto const pack_idx = lane_id / num_elems_per_pack;
          auto const elem_idx = lane_id % num_elems_per_pack;
          auto scale = extract_required_scale_format<kUseUE8M0>(
              ld_cg_global(src_scales + lane_id));
          recv_x_scales[token_idx * token_stride + pack_idx * pack_stride +
                        elem_idx] = scale;
        }
        if (lane_id + 32 < num_scales) {
          auto const pack_idx = (lane_id + 32) / num_elems_per_pack;
          auto const elem_idx = (lane_id + 32) % num_elems_per_pack;
          auto scale = extract_required_scale_format<kUseUE8M0>(
              ld_cg_global(src_scales + lane_id + 32));
          recv_x_scales[token_idx * token_stride + pack_idx * pack_stride +
                        elem_idx] = scale;
        }
      }
    }
    // if (blockIdx.x == 0 && threadIdx.x == 0)
    //   printf("[dispatch] RECV finished\n");
  }
}

void dispatch(void* packed_recv_x, void* packed_recv_x_scales,
              int* packed_recv_src_info, int64_t* packed_recv_layout_range,
              int* packed_recv_count, int* cumulative_local_expert_recv_stats,
              int64_t* dispatch_wait_recv_cost_stats, void* rdma_recv_x,
              int* rdma_recv_count, void* rdma_x, void const* x,
              int64_t const* topk_idx, int* next_clean, int* next_clean_second,
              int num_next_clean_int, int num_tokens, int hidden,
              int num_max_dispatch_tokens_per_rank, int num_topk,
              int num_experts, int rank, int num_ranks, bool use_fp8,
              bool round_scale, bool use_ue8m0, void* workspace,
              int num_device_sms, hipStream_t stream, int phases,
              uint64_t const* ring_addrs, int num_ring_addrs, int max_nvl_peers,
              int low_latency_buffer_idx, void** ipc_rdma_base_ptrs,
              void* rdma_buffer_ptr, void* atomic_buffer_ptr,
              int* rdma_recv_count_internode) {
  constexpr int kNumMaxTopK = 9;
  int const num_warp_groups = ceil_div(num_experts, num_device_sms);
  int const num_warps_per_group = 32 / num_warp_groups;
  EP_HOST_ASSERT(num_warp_groups > 0 and num_warps_per_group > 0);
  EP_HOST_ASSERT(kNumMaxTopK + 1 <= num_warp_groups * num_warps_per_group);

  auto const num_warps = num_warp_groups * num_warps_per_group;
  auto const num_sms = ceil_div(num_experts, num_warp_groups);
  EP_HOST_ASSERT(num_topk <= kNumMaxTopK);

  // Workspace checks
  auto atomic_counter_per_expert = static_cast<int*>(workspace);
  auto atomic_finish_counter_per_expert =
      atomic_counter_per_expert + num_experts;
  EP_HOST_ASSERT(num_experts * sizeof(int) * 2 <= NUM_WORKSPACE_BYTES);

  // FP8 checks
  if (use_ue8m0)
    EP_HOST_ASSERT(round_scale and "UE8M0 SF requires `round_scale=True`");

#define DISPATCH_LAUNCH_CASE(hidden)                                          \
  {                                                                           \
    auto dispatch_func = dispatch<false, false, hidden>;                      \
    if (use_fp8 and not use_ue8m0)                                            \
      dispatch_func = dispatch<true, false, hidden>;                          \
    if (use_fp8 and use_ue8m0) dispatch_func = dispatch<true, true, hidden>;  \
    LAUNCH_KERNEL(                                                            \
        &cfg, dispatch_func, packed_recv_x, packed_recv_x_scales,             \
        packed_recv_src_info, packed_recv_layout_range, packed_recv_count,    \
        cumulative_local_expert_recv_stats, dispatch_wait_recv_cost_stats,    \
        rdma_recv_x, rdma_recv_count, rdma_x, x, topk_idx,                    \
        atomic_counter_per_expert, atomic_finish_counter_per_expert,          \
        next_clean, next_clean_second, num_next_clean_int, num_tokens,        \
        num_max_dispatch_tokens_per_rank, num_topk, num_experts, rank,        \
        num_ranks, num_warp_groups, num_warps_per_group, round_scale, phases, \
        ring_addrs, num_ring_addrs, max_nvl_peers, low_latency_buffer_idx,    \
        ipc_rdma_base_ptrs, rdma_buffer_ptr, atomic_buffer_ptr,               \
        rdma_recv_count_internode);                                           \
  }                                                                           \
  break

  SETUP_LAUNCH_CONFIG(num_sms, num_warps * 32, stream);
  SWITCH_HIDDEN(DISPATCH_LAUNCH_CASE);
  auto err = hipGetLastError();
  if (err != hipSuccess) {
    printf("[dispatch] kernel launch error: %s\n", hipGetErrorString(err));
    fflush(stdout);
  }

#undef DISPATCH_LAUNCH_CASE
}

template <bool kUseLogFMT, int kHidden, int kNumMaxTopk>
__global__ __launch_bounds__(1024, 1) void combine(
    void* combined_x, void* rdma_recv_x, int* rdma_recv_flag, void* rdma_send_x,
    void const* x, int64_t const* topk_idx, float const* topk_weights,
    int const* src_info, int64_t const* layout_range,
    int64_t* combine_wait_recv_cost_stats, int* next_clean,
    int* next_clean_second, int num_next_clean_int, int* atomic_clean_flag,
    int num_combined_tokens, int hidden, int num_topk,
    int num_max_dispatch_tokens_per_rank, int num_experts, int rank,
    int num_ranks, int num_warp_groups, int num_warps_per_group, int phases,
    bool zero_copy, uint64_t const* ring_addrs, int num_ring_addrs,
    int max_nvl_peers, int low_latency_buffer_idx,
    void** ipc_rdma_base_ptrs = nullptr, void* rdma_buffer_ptr = nullptr,
    void* atomic_buffer_ptr = nullptr,
    int* rdma_recv_flag_internode = nullptr) {
  auto const sm_id = static_cast<int>(blockIdx.x);
  auto const num_sms = static_cast<int>(gridDim.x);
  auto const thread_id = static_cast<int>(threadIdx.x);
  auto const num_threads = static_cast<int>(blockDim.x);
  auto const warp_id = thread_id / 32, lane_id = get_lane_id();
  auto const num_local_experts = num_experts / num_ranks;
  auto const warp_group_id = warp_id / num_warps_per_group;
  auto const sub_warp_id = warp_id % num_warps_per_group;
  auto const responsible_expert_idx = sm_id * num_warp_groups + warp_group_id;
  // Data type staffs
  constexpr int kNumElemsPerInt4 = sizeof(int4) / sizeof(hip_bfloat16);
  constexpr int64_t hidden_bf16_int4 = kHidden / kNumElemsPerInt4;
  constexpr int kNumUnrolls = 4;
  constexpr int hidden_bf16_int4_pad =
      align(static_cast<int>(hidden_bf16_int4), 32 * kNumUnrolls);
  EP_STATIC_ASSERT(hidden_bf16_int4 % kNumUnrolls == 0, "Invalid hidden");
  EP_STATIC_ASSERT(kNumUnrolls == 1 or kNumUnrolls == 2 or kNumUnrolls == 4,
                   "Invalid unrolling factors");

  // Message package
  constexpr size_t num_bytes_per_slot = kHidden * sizeof(hip_bfloat16);
  EP_STATIC_ASSERT(num_bytes_per_slot % sizeof(int4) == 0,
                   "Invalid vectorization");

  // Sending phase
  if ((phases & LOW_LATENCY_SEND_PHASE) == 0) goto LOW_LATENCY_COMBINE_RECV;

  // Clean up next buffer
  if (sm_id == 0 and warp_group_id == 0 and sub_warp_id == 0) {
#pragma unroll
    for (int i = lane_id; i < num_next_clean_int; i += 32) {
      next_clean[i] = 0;
      next_clean_second[i] = 0;
    }

    // Notify before executing `int_p`
    __syncwarp();
    if (lane_id == 0) atomic_add_release_global(atomic_clean_flag, num_experts);
  }

  // Issue IBGDA sends
  if (responsible_expert_idx < num_experts) {
    auto const dst_rank = responsible_expert_idx / num_local_experts;
    auto const local_expert_idx = responsible_expert_idx % num_local_experts;
    auto const global_expert_idx = rank * num_local_experts + local_expert_idx;
    auto const layout =
        __ldg(layout_range + local_expert_idx * num_ranks + dst_rank);
    auto const local_x =
        static_cast<int4 const*>(x) + local_expert_idx * num_ranks *
                                          num_max_dispatch_tokens_per_rank *
                                          hidden_bf16_int4;
    auto const local_src_info = src_info + local_expert_idx * num_ranks *
                                               num_max_dispatch_tokens_per_rank;
    auto const rdma_send_x_vec = static_cast<uint8_t*>(rdma_send_x) +
                                 local_expert_idx * num_ranks *
                                     num_max_dispatch_tokens_per_rank *
                                     num_bytes_per_slot;

    // Unpack layout
    int offset, num_tokens_to_send;
    unpack2(layout, num_tokens_to_send, offset);

    // TMA stuffs
    constexpr int kNumTMABufferBytes = sizeof(int4) * 32 * kNumUnrolls;
    constexpr int kNumStages = 3;
    constexpr int kNumPrefetch = 1;
    EP_STATIC_ASSERT(kNumStages == 3 and kNumPrefetch == 1, "Invalid stages");

    extern __shared__ __align__(1024) uint8_t smem_buffer[];
    auto smem_ptr =
        smem_buffer + warp_id * kNumStages * (kNumTMABufferBytes + 16);
    uint32_t tma_phase[kNumStages] = {};
    auto tma_buffer = PatternVisitor([=](int const& i) {
      return reinterpret_cast<int4*>(smem_ptr + i * (kNumTMABufferBytes + 16));
    });
    auto tma_mbarrier = PatternVisitor([=](int const& i) {
      return reinterpret_cast<uint64_t*>(
          smem_ptr + i * (kNumTMABufferBytes + 16) + kNumTMABufferBytes);
    });
    EP_STATIC_ASSERT(kNumUnrolls * kNumStages <= 12,
                     "TMA buffer size exceed limit");

    // Initialize m-barriers
    if (lane_id < kNumStages) {
      mbarrier_init(tma_mbarrier[lane_id], 1);
      fence_view_async_shared();
      fence_barrier_init();
    }
    __syncwarp();

    constexpr int kNumIters = hidden_bf16_int4_pad / (32 * kNumUnrolls);
    auto tma_load_and_arrive = [&](int const& stage_idx, int4 const* gmem_ptr,
                                   int const& num_bytes) {
      tma_load_1d(tma_buffer[stage_idx], gmem_ptr, tma_mbarrier[stage_idx],
                  num_bytes);
      mbarrier_arrive_and_expect_tx(tma_mbarrier[stage_idx], num_bytes);
    };
    auto get_num_tma_bytes = [&](int const& offset_int4) {
      return min(
          kNumTMABufferBytes,
          static_cast<int>((hidden_bf16_int4 - offset_int4) * sizeof(int4)));
    };

    // Issue IBGDA send
    for (int token_idx = offset + sub_warp_id;
         token_idx < offset + num_tokens_to_send;
         token_idx += num_warps_per_group) {
      auto const x_int4 = local_x + token_idx * hidden_bf16_int4;
      auto const rdma_send_type_row = reinterpret_cast<int*>(
          rdma_send_x_vec + token_idx * num_bytes_per_slot);
      auto const rdma_send_x_vec_row =
          reinterpret_cast<uint8_t*>(rdma_send_type_row);

      auto const src_idx =
          __shfl_sync(0xffffffff, __ldg(local_src_info + token_idx), 0);
      auto const buf_ptr = reinterpret_cast<int64_t>(rdma_send_x_vec_row);
      auto const dst_ptr =
          reinterpret_cast<uint64_t>(rdma_recv_x) +
          (global_expert_idx * num_max_dispatch_tokens_per_rank + src_idx) *
              num_bytes_per_slot;
      // TODO(yihan): Mark here for future debugging check.
      // ORIGINAL CODE: Calculate absolute destination address using local
      // rdma_recv_x base auto const dst_ptr =
      //     reinterpret_cast<uint64_t>(rdma_recv_x) +
      //     (global_expert_idx * num_max_dispatch_tokens_per_rank + src_idx) *
      //         num_bytes_per_slot;

      // Use IPC for intra-node P2P mapping when available
      auto const dst_p2p_ptr =
          ipc_rdma_base_ptrs
              ? uccl::get_ipc_p2p_ptr(dst_ptr, ipc_rdma_base_ptrs, rank,
                                      dst_rank, max_nvl_peers, 0)
              : 0;

      if (not zero_copy or dst_p2p_ptr != 0) {
        // Read from `cpy_src_int4_ptr` and copy into `cpy_dst_int4_ptr`
        auto const cpy_src_int4_ptr =
            zero_copy ? reinterpret_cast<int4*>(buf_ptr) : x_int4;
        auto const cpy_dst_int4_ptr =
            dst_p2p_ptr == 0 ? reinterpret_cast<int4*>(buf_ptr)
                             : reinterpret_cast<int4*>(dst_p2p_ptr);

        // Prefetch
        if (elect_one_sync(lane_id))
          tma_load_and_arrive(0, cpy_src_int4_ptr, get_num_tma_bytes(0));
        __syncwarp();

#pragma unroll
        for (int i = lane_id * kNumUnrolls, iter_idx = 0;
             i < hidden_bf16_int4_pad; i += 32 * kNumUnrolls, ++iter_idx) {
          // Read
          int4 int4_values[kNumUnrolls] = {0};
          auto uint32_values = reinterpret_cast<uint32_t*>(int4_values);

          // Load the next iteration
          // TODO: try `elect_one_sync`
          int const& stage_idx = iter_idx % kNumStages;
          int const& next_stage_idx = (iter_idx + 1) % kNumStages;
          tma_store_wait<kNumStages - kNumPrefetch - 1>();
          if (iter_idx + 1 < kNumIters and elect_one_sync(lane_id)) {
            auto const& offset_int4 = i + 32 * kNumUnrolls;
            tma_load_and_arrive(next_stage_idx, cpy_src_int4_ptr + offset_int4,
                                get_num_tma_bytes(offset_int4));
          }
          __syncwarp();

          // Wait the current TMA arrival
          mbarrier_wait(tma_mbarrier[stage_idx], tma_phase[stage_idx]);
          auto const& uint32_buffer = reinterpret_cast<uint32_t*>(
              tma_buffer[stage_idx] + lane_id * kNumUnrolls);

          // Simulated cast
          if constexpr (kUseLogFMT) {
            constexpr float kThreshold = 1;
            constexpr float kMinClip = 32;  // `== log_2(2 ^ (2 ^ 5))`
            constexpr int kNumBits = 10;
            constexpr int kNumValues = 1 << (kNumBits - 1);
            EP_STATIC_ASSERT(kHidden % (kNumElemsPerInt4 * 32) == 0 and
                                 kNumElemsPerInt4 == 8,
                             "Invalid hidden");

            // Local log amax
            float log_abs_values[kNumElemsPerInt4 * kNumUnrolls], log_amax,
                log_amin, amax;
            auto log_aminmax = [&](int const& j, float const& value) {
              log_abs_values[j] = log2f_approx(fabsf(value));
              amax = j == 0 ? value : fmaxf(amax, fabsf(value));
              log_amax = j == 0 ? log_abs_values[j]
                                : fmaxf(log_amax, log_abs_values[j]);
              log_amin = value != 0
                             ? (j == 0 ? log_abs_values[j]
                                       : fminf(log_amin, log_abs_values[j]))
                             : log_amin;
            };
#pragma unroll
            for (int k = 0; k < kNumUnrolls * 4; ++k) {
              uint32_values[k] = uint32_buffer[k ^ (lane_id * kNumUnrolls / 8)];
              auto bf162_values =
                  *reinterpret_cast<__hip_bfloat162*>(uint32_values + k);
              auto float2_values = __bfloat1622float2(bf162_values);
              log_aminmax(k * 2, float2_values.x);
              log_aminmax(k * 2 + 1, float2_values.y);
            }

            // Reduce per 128 channels
            amax = warp_reduce_max<(16 / kNumUnrolls)>(amax);
            log_amax = warp_reduce_max<(16 / kNumUnrolls)>(log_amax);
            log_amin = fmaxf(warp_reduce_min<(16 / kNumUnrolls)>(log_amin),
                             log_amax - kMinClip);

            auto const step =
                (log_amax - log_amin) / static_cast<float>(kNumValues - 2);
            auto const step_inv = 1.0f / step;
            auto const rounding =
                2.0f -
                log2f_approx((1.0f + exp2f_approx(step)) * 0.5f) * step_inv;

            // Use LogFMT only with `amax <= kThreshold` (maybe not all
            // quarter-warps)
            if (amax <= kThreshold and log_amin < log_amax) {
              // Transform
              auto transform = [=](float const& log_abs_value) -> hip_bfloat16 {
                auto const encoded =
                    floorf((log_abs_value - log_amin) * step_inv + rounding);
                auto const decoded =
                    exp2f_approx((encoded - 1) * step + log_amin);
                return decoded;
              };
#pragma unroll
              for (int k = 0; k < kNumUnrolls * 4; ++k) {
                auto bf162_pack =
                    __hip_bfloat162(transform(log_abs_values[k * 2]),
                                   transform(log_abs_values[k * 2 + 1]));
                auto uint32_pack = *reinterpret_cast<uint32_t*>(&bf162_pack);
                uint32_buffer[k ^ (lane_id * kNumUnrolls / 8)] =
                    (uint32_values[k] & 0x80008000) | uint32_pack;
              }
            }
            tma_store_fence();
          }
          __syncwarp();

          // Store
          if (elect_one_sync(lane_id))
            tma_store_1d(tma_buffer[stage_idx], cpy_dst_int4_ptr + i,
                         get_num_tma_bytes(i));
          __syncwarp();
        }
      }
      // Flush all stores
      tma_store_wait();
      __syncwarp();

      // Issue RDMA only if we couldn't use IPC
      // NOTES: for zero-copy mode, we assume the data is already in the send
      // buffer
      if (dst_p2p_ptr == 0) {
        __threadfence_system();
        nvshmemi_ibgda_put_nbi_warp(
            dst_ptr - reinterpret_cast<uint64_t>(rdma_buffer_ptr), buf_ptr,
            hidden * sizeof(hip_bfloat16), dst_rank,
            /*warp_id=*/global_expert_idx,  // NOTE(Yang): for selecting rb.
            // NOTE(Ziming): this is global_expert_idx because destination is
            // indexed by global_expert_idx
            lane_id, token_idx - offset, ring_addrs, num_ring_addrs, true,
            low_latency_buffer_idx);
      }
    }

    // Put the finishing flag
    EP_DEVICE_ASSERT(num_warps_per_group > 1 and num_warp_groups < 16);
    sync_barrier(warp_group_id + 1, num_warps_per_group * 32);
    if (sub_warp_id == 1 and lane_id == 0) {
      while (ld_acquire_global(atomic_clean_flag) == 0)
        ;
      // Calculate offset from data buffer to flag buffer (similar to dispatch
      // phase) rdma_recv_flag_internode corresponds to
      // combine_rdma_recv_flag_buffer We need to calculate the offset from
      // rdma_recv_x (data buffer) to the flag buffer Try to use IPC for
      // intra-node atomic operations
      auto dst_ptr =
          reinterpret_cast<uint64_t>(rdma_recv_flag + global_expert_idx);
      auto dst_ptr_internode = reinterpret_cast<uint64_t>(
          rdma_recv_flag_internode + global_expert_idx);
      auto dst_p2p_ptr =
          ipc_rdma_base_ptrs
              ? uccl::get_ipc_p2p_ptr(dst_ptr, ipc_rdma_base_ptrs, rank,
                                      dst_rank, max_nvl_peers, 0)
              : 0;
      if (dst_p2p_ptr != 0) {
        // Intra-node: use direct atomic operation
        st_release_sys_global(reinterpret_cast<int*>(dst_p2p_ptr), 1);
      } else {
        // Inter-node or no IPC: use IBGDA atomic
        // NOTE(MaoZiming): Without ibgda, we can only use atomic add
        // Pass offset to CPU proxy for atomic operation (similar to dispatch
        // phase)
        uccl::nvshmemi_ibgda_amo_nonfetch_add(
            dst_ptr_internode - reinterpret_cast<uint64_t>(atomic_buffer_ptr),
            num_tokens_to_send /* Will be changed to 1 in the proxy */,
            dst_rank,
            /*qp_id=*/-1,                   // NOTE(Yang): not used.
            /*warp_id=*/global_expert_idx,  // NOTE(Yang): for selecting rb.
            false, ring_addrs, num_ring_addrs, true, low_latency_buffer_idx);
      }
      atomic_add_release_global(atomic_clean_flag, -1);
    }
    __syncwarp();
  }

// Receiving phase
LOW_LATENCY_COMBINE_RECV:
  if ((phases & LOW_LATENCY_RECV_PHASE) == 0) {
    // if (blockIdx.x == 0 && threadIdx.x == 0)
    //   printf("[combine] SEND finished\n");
    return;
  }
  // Wait all ranks to arrive
  if (responsible_expert_idx < num_experts) {
    EP_DEVICE_ASSERT(num_warps_per_group > 1);
    if (sub_warp_id == 0 and lane_id == 0) {
      auto const src_rank = responsible_expert_idx / num_local_experts;
      auto start_time = clock64();
      while ((src_rank / max_nvl_peers == rank / max_nvl_peers) &&
             ld_acquire_sys_global(rdma_recv_flag + responsible_expert_idx) ==
                 0)
        ;
      while ((src_rank / max_nvl_peers != rank / max_nvl_peers) &&
             ld_acquire_sys_global(rdma_recv_flag_internode +
                                   responsible_expert_idx) == 0)
        ;

      if (src_rank / max_nvl_peers == rank / max_nvl_peers) {
        if (ld_acquire_sys_global(rdma_recv_flag_internode +
                                  responsible_expert_idx) != 0) {
          printf(
              "Same node but rdma_recv_flag_internode is not zero! src_rank: "
              "%d, rank: %d, max_nvl_peers: %d\n",
              src_rank, rank, max_nvl_peers);
          assert(false);
        }
      }
      if (src_rank / max_nvl_peers != rank / max_nvl_peers) {
        if (ld_acquire_sys_global(rdma_recv_flag + responsible_expert_idx) !=
            0) {
          printf(
              "Different node but rdma_recv_flag is not zero! src_rank: %d, "
              "rank: %d, max_nvl_peers: %d\n",
              src_rank, rank, max_nvl_peers);
          assert(false);
        }
      }

      auto wait_recv_cost = clock64() - start_time;
      if (combine_wait_recv_cost_stats != nullptr) {
        auto const& src_rank = responsible_expert_idx / num_local_experts;
        atomicAdd(reinterpret_cast<unsigned long long*>(
                      combine_wait_recv_cost_stats + src_rank),
                  wait_recv_cost);
      }
    }
  }
  cg::this_grid().sync();

  // Reduce tokens
  EP_DEVICE_ASSERT(num_topk <= 32);
  EP_STATIC_ASSERT(kHidden % (32 * kNumElemsPerInt4) == 0,
                   "Invalid vectorization");
  for (int hidden_idx = thread_id; hidden_idx < hidden_bf16_int4;
       hidden_idx += num_threads) {
    for (int token_idx = sm_id; token_idx < num_combined_tokens;
         token_idx += num_sms) {
      // Read top-k indices and weights
      int reg_topk_idx[kNumMaxTopk];
      float reg_topk_weights[kNumMaxTopk];
#pragma unroll
      for (int i = 0; i < num_topk; ++i) {
        reg_topk_idx[i] =
            static_cast<int>(__ldg(topk_idx + token_idx * num_topk + i));
        reg_topk_weights[i] = __ldg(topk_weights + token_idx * num_topk + i);
      }

      float combined_values[kNumElemsPerInt4] = {0.0f};
#pragma unroll
      for (int i = 0; i < num_topk; ++i)
        if (reg_topk_idx[i] >= 0) {
          // Read from sources
          auto rdma_buffer_type = reinterpret_cast<int const*>(
              static_cast<uint8_t*>(rdma_recv_x) +
              (reg_topk_idx[i] * num_max_dispatch_tokens_per_rank + token_idx) *
                  num_bytes_per_slot);
          auto rdma_buffer_row =
              reinterpret_cast<uint8_t const*>(rdma_buffer_type);

          // Reduce
          auto x_vec = ld_cg_global(
              reinterpret_cast<int4 const*>(rdma_buffer_row) + hidden_idx);
          auto const x_bf16 = reinterpret_cast<hip_bfloat16*>(&x_vec);
#pragma unroll
          for (int j = 0; j < kNumElemsPerInt4; ++j)
            combined_values[j] +=
                static_cast<float>(x_bf16[j]) * reg_topk_weights[i];
        }

      // Write results
      int4& combined_int4 = *reinterpret_cast<int4*>(combined_values);
      auto combined_bf16 = reinterpret_cast<hip_bfloat16*>(&combined_values);
#pragma unroll
      for (int j = 0; j < kNumElemsPerInt4; ++j)
        combined_bf16[j] = static_cast<hip_bfloat16>(combined_values[j]);
      (static_cast<int4*>(combined_x) +
       token_idx * hidden_bf16_int4)[hidden_idx] = combined_int4;
    }

    // if (blockIdx.x == 0 && threadIdx.x == 0)
    //   printf("[combine] RECV finished\n");
  }
}

void combine(void* combined_x, void* rdma_recv_x, int* rdma_recv_flag,
             void* rdma_send_x, void const* x, int64_t const* topk_idx,
             float const* topk_weights, int const* src_info,
             int64_t const* layout_range, int64_t* combine_wait_recv_cost_stats,
             int* next_clean, int* next_clean_second, int num_next_clean_int,
             int num_combined_tokens, int hidden,
             int num_max_dispatch_tokens_per_rank, int num_topk,
             int num_experts, int rank, int num_ranks, bool use_logfmt,
             void* workspace, int num_device_sms, hipStream_t stream,
             int phases, bool zero_copy, uint64_t const* ring_addrs,
             int num_ring_addrs, int max_nvl_peers, int low_latency_buffer_idx,
             void** ipc_rdma_base_ptrs, void* rdma_buffer_ptr,
             void* atomic_buffer_ptr, int* rdma_recv_flag_internode) {
  constexpr int kNumMaxTopk = 9;
  int const num_warp_groups = ceil_div(num_experts, num_device_sms);
  int const num_warps_per_group = 32 / num_warp_groups;
  EP_HOST_ASSERT(num_warp_groups > 0 and num_warps_per_group > 0);

  auto const num_warps = num_warp_groups * num_warps_per_group;
  auto const num_sms = ceil_div(num_experts, num_warp_groups);

  // Check workspace
  auto atomic_clean_flag = static_cast<int*>(workspace);
  EP_HOST_ASSERT(sizeof(int) <= NUM_WORKSPACE_BYTES);
  EP_HOST_ASSERT(num_topk <= kNumMaxTopk);

  // Online cast cannot use zero-copy
  EP_HOST_ASSERT(not(zero_copy and use_logfmt));

  constexpr int kNumTMABytesPerWarp = 12 * (512 + 16);
  int const smem_size = kNumTMABytesPerWarp * num_warps;
  // printf("Combine launched\n");

#define COMBINE_LAUNCH_CASE(hidden)                                            \
  {                                                                            \
    auto combine_func = use_logfmt ? combine<true, hidden, kNumMaxTopk>        \
                                   : combine<false, hidden, kNumMaxTopk>;      \
    SET_SHARED_MEMORY_FOR_TMA(combine_func);                                   \
    LAUNCH_KERNEL(&cfg, combine_func, combined_x, rdma_recv_x, rdma_recv_flag, \
                  rdma_send_x, x, topk_idx, topk_weights, src_info,            \
                  layout_range, combine_wait_recv_cost_stats, next_clean,      \
                  next_clean_second, num_next_clean_int, atomic_clean_flag,    \
                  num_combined_tokens, hidden, num_topk,                       \
                  num_max_dispatch_tokens_per_rank, num_experts, rank,         \
                  num_ranks, num_warp_groups, num_warps_per_group, phases,     \
                  zero_copy, ring_addrs, num_ring_addrs, max_nvl_peers,        \
                  low_latency_buffer_idx, ipc_rdma_base_ptrs, rdma_buffer_ptr, \
                  atomic_buffer_ptr, rdma_recv_flag_internode);                \
  }                                                                            \
  break

  SETUP_LAUNCH_CONFIG(num_sms, num_warps * 32, stream);
  SWITCH_HIDDEN(COMBINE_LAUNCH_CASE);
  auto err = hipGetLastError();
  if (err != hipSuccess) {
    printf("[combine] kernel launch error: %s\n", hipGetErrorString(err));
    fflush(stdout);
  }
#undef COMBINE_LAUNCH_CASE
}

}  // namespace internode_ll
}  // namespace uccl