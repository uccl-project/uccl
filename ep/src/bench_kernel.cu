#include "hip/hip_runtime.h"
#include "bench_kernel.cuh"
#include "common.hpp"
#include "fifo_device.hpp"
#include "ring_buffer.cuh"
#include <stdint.h>
#include <stdio.h>

__global__ void gpu_issue_batched_commands(DeviceToHostCmdBuffer* rbs) {
  int const bid = blockIdx.x;
  int const tid = threadIdx.x;
  int const num_threads = blockDim.x;

  if (tid == 0) {
    printf("Device Block %d: Scheduled with %d threads\n", bid, num_threads);
  }
  __syncthreads();

  auto rb = &rbs[bid];
  uint32_t completed = tid;

  __shared__ unsigned long long cycle_accum_smem;
  __shared__ unsigned int op_count_smem;
  __shared__ uint64_t shared_cycle_start;
  __shared__ unsigned long long start_cycle_smem[kQueueSize];

#define kInflightSlotSize (kMaxInflight * kNumThPerBlock)
#define kInflightSlotMask (kInflightSlotSize - 1)
  uint64_t inflight_slots[kInflightSlotSize];

  if (tid == 0) {
    cycle_accum_smem = 0ull;
    op_count_smem = 0u;
    shared_cycle_start = 0;
    rb->cycle_start = 0;
  }
  __syncthreads();

  // Each thread dispatches its own commands with stride
  for (uint32_t it = tid; it < kIterations; it += num_threads) {
    uint64_t cur_tail = rb->volatile_tail();

    // Check if there are any completed commands
    while (completed < cur_tail) {
      if (completed >= kWarmupOps) {
        unsigned long long t1 = clock64();
        uint64_t inflight_slot =
            inflight_slots[(completed / kNumThPerBlock) & kInflightSlotMask];
        unsigned long long t0 = start_cycle_smem[inflight_slot & kQueueMask];
        unsigned long long cycles = t1 - t0;
        atomicAdd((unsigned long long*)&cycle_accum_smem, cycles);
        atomicAdd(&op_count_smem, 1u);
        atomicCAS((unsigned long long*)&shared_cycle_start, 0ULL, t1);
      }
      completed += num_threads;
    }

    // Check global ring buffer state and wait if necessary
    while (true) {
      uint64_t cur_head = rb->head;
      cur_tail = rb->volatile_tail();
      uint64_t inflight = cur_head - cur_tail;

      if (inflight < kInflightSlotSize) {
        // Record start time
        unsigned long long t0 = clock64();
        uint64_t my_slot = cur_head;
        // Create the command
        TransferCmd cmd{.cmd_type = make_cmd_type(CmdType::WRITE, false, 0),
                        .dst_rank = 1,
                        .bytes_and_val = 7168,
                        .req_rptr = 0,
                        .req_lptr = 0};

        // Space available, atomically reserve and commit
        rb->atomic_set_and_commit(cmd, &my_slot);
        start_cycle_smem[my_slot & kQueueMask] = t0;
        inflight_slots[(it / kNumThPerBlock) & kInflightSlotMask] = my_slot;
        break;
      } else {
        // Otherwise, it is gonna hang here.
        __nanosleep(64);
      }
    }
  }

  // Polling the remaining requests.
  while (completed < kIterations) {
    uint64_t cur_tail = rb->volatile_tail();
    while (completed < cur_tail) {
      if (completed >= kWarmupOps) {
        unsigned long long t1 = clock64();
        uint64_t inflight_slot =
            inflight_slots[(completed / kNumThPerBlock) & kInflightSlotMask];
        unsigned long long t0 = start_cycle_smem[inflight_slot & kQueueMask];
        unsigned long long cycles = t1 - t0;
        atomicAdd((unsigned long long*)&cycle_accum_smem, cycles);
        atomicAdd(&op_count_smem, 1u);
        atomicCAS((unsigned long long*)&shared_cycle_start, 0ULL, t1);
      }
      completed += num_threads;
    }
  }

  __syncthreads();

  if (tid == 0) {
    rb->cycle_start = shared_cycle_start;
    rb->cycle_accum = cycle_accum_smem;
    rb->op_count = op_count_smem;
    rb->cycle_end = clock64();
    printf("Device Block %d done (%d threads, measured %u ops)\n", bid,
           num_threads, op_count_smem);
  }
}

// ============================================================================
// Ring buffer kernel launcher
// ============================================================================

hipError_t launch_gpu_issue_batched_commands_shim(int blocks,
                                                   int threads_per_block,
                                                   size_t shmem_bytes,
                                                   hipStream_t stream,
                                                   DeviceToHostCmdBuffer* rbs) {
  gpu_issue_batched_commands<<<blocks, threads_per_block, shmem_bytes,
                               stream>>>(rbs);
  return hipGetLastError();
}

// ============================================================================
// FIFO-based GPU kernel
// ============================================================================

// FIFO-based GPU kernel - each block uses its own FIFO
// Implements kMaxInflight limiting with proper polling and latency measurement
__global__ void gpu_issue_batched_commands_fifo(
    mscclpp::FifoDeviceHandle* fifos, uint64_t* cycle_start_out,
    uint64_t* cycle_end_out, uint64_t* cycle_accum_out,
    uint32_t* op_count_out) {
  int const bid = blockIdx.x;
  int const tid = threadIdx.x;
  int const num_threads = blockDim.x;

  if (tid == 0) {
    printf("Device Block %d: Scheduled with %d threads (FIFO mode)\n", bid,
           num_threads);
  }

  // Get this block's FIFO (shared by all threads)
  mscclpp::FifoDeviceHandle& fifo = fifos[bid];

  // Track completed operations and latency metrics (per-thread)
  uint32_t completed = 0;

  __shared__ unsigned long long cycle_accum_smem;
  __shared__ unsigned int op_count_smem;
  extern __shared__ unsigned long long start_cycle_smem[];
  if (tid == 0) {
    cycle_accum_smem = 0ull;
    op_count_smem = 0u;
  }
  __syncthreads();

  // Track in-flight requests using circular buffer (per-thread)
  // When we reach kMaxInflight, we poll the oldest to maintain the limit
  uint64_t head_buffer[kMaxInflight];
  uint32_t head_write_idx = 0;
  uint32_t head_read_idx = 0;
  uint32_t inflight_count = 0;
  uint64_t block_cycle_start = 0;

  // Each thread processes kIterations/num_threads operations
  int my_iterations = (kIterations + num_threads - 1) / num_threads;

  for (int local_it = 0; local_it < my_iterations; ++local_it) {
    int it = tid + local_it * num_threads;
    if (it >= kIterations) break;

    unsigned long long t0 = clock64();
    start_cycle_smem[it & kQueueMask] = t0;

    int message_idx = it + 1;

    // Push to FIFO
    mscclpp::ProxyTrigger trigger;
    trigger.fst = (static_cast<uint64_t>(bid + 1) << 32) |
                  (static_cast<uint64_t>(CmdType::WRITE) & 0xFFFFFFFF);
    trigger.snd = message_idx;
    uint64_t head = fifo.push(trigger);

    // Store head in circular buffer for tracking
    head_buffer[head_write_idx] = head;
    head_write_idx = (head_write_idx + 1) % kMaxInflight;
    inflight_count++;

    // Once we reach kMaxInflight, poll the oldest request to keep under limit
    if (inflight_count >= kMaxInflight) {
      uint64_t oldest_head = head_buffer[head_read_idx];
      head_read_idx = (head_read_idx + 1) % kMaxInflight;

      // Wait for the oldest request to be consumed by host proxy
      fifo.sync(oldest_head, -1);

      // Measure latency for completed operation
      int abs_it = tid + completed * num_threads;
      if (abs_it >= kWarmupOps && abs_it < kIterations) {
        unsigned long long t1 = clock64();
        unsigned long long cycles = t1 - start_cycle_smem[abs_it & kQueueMask];
        atomicAdd((unsigned long long*)&cycle_accum_smem, cycles);
        atomicAdd(&op_count_smem, 1u);
        if (block_cycle_start == 0) {
          block_cycle_start = t1;
        }
      }
      completed++;
      inflight_count--;
    }
  }

  // Wait for all remaining in-flight operations to complete
  while (inflight_count > 0) {
    uint64_t oldest_head = head_buffer[head_read_idx];
    head_read_idx = (head_read_idx + 1) % kMaxInflight;

    fifo.sync(oldest_head, -1);
    inflight_count--;

    int abs_it = tid + completed * num_threads;
    if (abs_it < kIterations && abs_it >= kWarmupOps) {
      unsigned long long t1 = clock64();
      unsigned long long cycles = t1 - start_cycle_smem[abs_it & kQueueMask];
      atomicAdd((unsigned long long*)&cycle_accum_smem, cycles);
      atomicAdd(&op_count_smem, 1u);
    }
    completed++;
  }

  __syncthreads();

  if (tid == 0) {
    if (cycle_start_out) cycle_start_out[bid] = block_cycle_start;
    if (cycle_end_out) cycle_end_out[bid] = clock64();
    if (cycle_accum_out) cycle_accum_out[bid] = cycle_accum_smem;
    if (op_count_out) op_count_out[bid] = op_count_smem;

    printf(
        "Device Block %d done (%d threads pushed %d operations, measured %u "
        "ops)\n",
        bid, num_threads, kIterations, op_count_smem);
  }
}

// Launcher function implementation
hipError_t launch_gpu_issue_batched_commands_fifo(
    int blocks, int threads_per_block, size_t shmem_bytes, hipStream_t stream,
    mscclpp::FifoDeviceHandle* d_fifos, uint64_t* cycle_start,
    uint64_t* cycle_end, uint64_t* cycle_accum, uint32_t* op_count) {
  gpu_issue_batched_commands_fifo<<<blocks, threads_per_block, shmem_bytes,
                                    stream>>>(d_fifos, cycle_start, cycle_end,
                                              cycle_accum, op_count);

  return hipGetLastError();
}
