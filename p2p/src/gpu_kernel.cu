#include "hip/hip_runtime.h"
#include "common.hpp"
#include "gpu_kernel.cuh"
#include "ring_buffer.cuh"
#include <cuda_pipeline.h>
#include <stdint.h>
#include <stdio.h>

__global__ void gpu_issue_batched_commands(
    RingBuffer<TransferCmd, FlowDirection::DeviceToHost, kQueueSize>* rbs) {
  int const bid = blockIdx.x;
  int const tid = threadIdx.x;
  RingBuffer<TransferCmd, FlowDirection::DeviceToHost, kQueueSize>* rb =
      &rbs[bid];
  if (tid != 0) {
    return;
  }
  printf("Device Block %d: Scheduled\n", bid);

#ifdef MEASURE_PER_OP_LATENCY
  uint32_t complete = 0;
  __shared__ unsigned long long cycle_accum_smem;
  __shared__ unsigned int op_count_smem;
  cycle_accum_smem = 0ull;
  op_count_smem = 0u;
#endif

  extern __shared__ unsigned long long start_cycle_smem[];
  bool print_warmup_exit = true;

  rb->cycle_start = 0;
  for (int it = 0; it < kIterations;) {
    uint64_t my_hdr;
    uint64_t cur_tail;

#ifdef MEASURE_PER_OP_LATENCY
    // if (complete < my_hdr + todo) {
    uint32_t cidx = complete & kQueueMask;
    cur_tail = rb->volatile_tail();
    if (complete < cur_tail) {
      // __threadfence_system();
      for (int i = complete; i < cur_tail; ++i) {
        if (rb->buf[cidx].cmd != 0) {
          printf(
              "Device Block %d: Error at complete %u, rb->tail:%lu, expected "
              "0, got %llu\n",
              bid, complete, rb->tail, rb->buf[cidx].cmd);
          return;
        }
        if (complete >= kWarmupOps) {
          unsigned long long t1 = clock64();
          unsigned long long cycles = t1 - start_cycle_smem[cidx];
          cycle_accum_smem += cycles;
          op_count_smem++;
          if (rb->cycle_start == 0) {
            rb->cycle_start = t1;
          }
        }
      }
      complete = cur_tail;
    }  // else {
       // break;
       // }
       // }
#endif

    unsigned int todo =
        (it + kBatchSize <= kIterations) ? kBatchSize : (kIterations - it);

    // Dynamically send the number of todos to send.
    // while (true) {
    uint64_t cur_head = rb->head;
    cur_tail = rb->volatile_tail();
    uint64_t free_slots = kMaxInflight - (cur_head - cur_tail);

    if (free_slots >= todo) {
      // rb->head = cur_head + todo;
      my_hdr = cur_head;
      // break;
    } else if (free_slots >= 1) {
      // rb->head = cur_head + free_slots;
      my_hdr = cur_head;
      todo = free_slots;
      // break;
    } else {
      continue;
    }
    /* Spin */
    // }

    for (int i = 0; i < todo; ++i) {
      uint32_t idx = (my_hdr + i) & kQueueMask;
      unsigned long long t0 = clock64();
      rb->buf[idx].cmd = (static_cast<uint64_t>(bid) << 32) | (it + i + 1);
      rb->buf[idx].dst_rank = bid;
      rb->buf[idx].dst_gpu = 0;
      rb->buf[idx].src_ptr =
          reinterpret_cast<void*>(static_cast<uintptr_t>(it + i + 1));
      rb->buf[idx].bytes = kObjectSize;
      start_cycle_smem[idx] = t0;
    }
    rb->commit();
    rb->head = my_hdr + todo;

    it += todo;
    if (complete > kWarmupOps) {
      if (print_warmup_exit) {
        printf("Device Block %d: Exiting warmup phase\n", bid);
        print_warmup_exit = false;
      }
    }
  }

#ifdef MEASURE_PER_OP_LATENCY
  while (complete < kIterations) {
    // while (complete >= ld_volatile(&rb->tail)) { /* spin */
    // }
    if (complete >= kWarmupOps && complete < ld_volatile(&rb->tail)) {
      unsigned long long t1 = clock64();
      cycle_accum_smem += (t1 - start_cycle_smem[complete & kQueueMask]);
      ++op_count_smem;
    } else {
      continue;
    }
    ++complete;
  }

  rb->cycle_accum = cycle_accum_smem;
  rb->op_count = op_count_smem;
#endif
  rb->cycle_end = clock64();
  printf("Device Block %d done\n", bid);
}