#include "hip/hip_runtime.h"
#include "common.hpp"
#include "gpu_kernel.cuh"
#include "proxy.hpp"
#include "rdma.hpp"
#include "ring_buffer.cuh"
#include <chrono>
#include <cstdlib>
#include <iostream>
#include <thread>
#include <vector>

inline void** allocate_memory_for_gpudirect() {
  size_t bytes_per_block = kObjectSize * kBatchSize;
  size_t total_bytes = bytes_per_block * kNumThBlocks;
  void* d_big = nullptr;
  hipMalloc(&d_big, total_bytes);

  std::vector<void*> h_ptrs(kNumThBlocks);
  for (int b = 0; b < kNumThBlocks; ++b)
    h_ptrs[b] = static_cast<char*>(d_big) + b * bytes_per_block;

  void** d_ptrs = nullptr;
  hipMalloc(&d_ptrs, sizeof(void*) * kNumThBlocks);
  hipMemcpy(d_ptrs, h_ptrs.data(), sizeof(void*) * kNumThBlocks,
             hipMemcpyHostToDevice);

  return d_ptrs;
}

int main(int argc, char** argv) {
  if (argc < 3) {
    std::cerr << "Usage: ./benchmark_remote <rank> <peer_ip>\n";
    return 1;
  }
  int rank = std::atoi(argv[1]);
  char const* peer_ip = argv[2];

  if (!GdrSupportInitOnce()) {
    printf(
        "Error: GPUDirect RDMA module is not loaded. Please load "
        "nvidia_peermem or nv_peer_mem!\n");
    exit(1);
  }

  hipStream_t stream1;
  hipStreamCreate(&stream1);
  cudaCheckErrors("hipStreamCreate failed");

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  printf("clock rate: %d kHz\n", prop.clockRate);

  RingBuffer* rbs;
  hipHostAlloc(&rbs, sizeof(RingBuffer) * kNumThBlocks, hipHostMallocMapped);
  for (int i = 0; i < kNumThBlocks; ++i) {
    rbs[i].head = 0;
    rbs[i].tail = 0;
    for (uint32_t j = 0; j < kQueueSize; ++j) {
      rbs[i].buf[j].cmd = 0;
    }
  }

  void** d_ptrs = allocate_memory_for_gpudirect();
  printf("Allocated %d bytes for %d blocks of %d bytes each\n",
         kObjectSize * kBatchSize * kNumThBlocks, kNumThBlocks,
         kObjectSize * kBatchSize);
  printf("d_ptrs address: %p\n", d_ptrs);

  size_t total_size = kObjectSize * kBatchSize * kNumThBlocks;
  void* gpu_buffer = nullptr;
  hipMalloc(&gpu_buffer, total_size);

  RDMAConnectionInfo local_info, remote_info;
  setup_rdma(gpu_buffer, total_size, &local_info, rank);

  modify_qp_to_init();

  printf("Local RDMA info: addr=0x%lx, rkey=0x%x\n", local_info.addr,
         local_info.rkey);
  exchange_connection_info(rank, peer_ip, &local_info, &remote_info);
  printf("Exchanged remote_addr: 0x%lx, remote_rkey: 0x%x\n", remote_info.addr,
         remote_info.rkey);

  modify_qp_to_rtr(&remote_info);
  modify_qp_to_rts(&local_info);

  remote_addr = remote_info.addr;
  remote_rkey = remote_info.rkey;

  if (rank == 0) {
    rdma_write_stub(gpu_buffer, total_size);
    printf("RDMA write stub completed\n");

    poll_completion();
    printf("Polling completions done\n");

    // Launch one CPU polling thread per block
    std::vector<std::thread> cpu_threads;
    for (int i = 0; i < kNumThBlocks; ++i) {
      cpu_threads.emplace_back(cpu_consume, &rbs[i], i, gpu_buffer, total_size,
                               rank);
    }
    auto t0 = std::chrono::high_resolution_clock::now();
    size_t shmem_bytes = kQueueSize * sizeof(unsigned long long);
    gpu_issue_batched_commands<<<kNumThBlocks, kNumThPerBlock, shmem_bytes,
                                 stream1>>>(rbs, d_ptrs);
    cudaCheckErrors("gpu_issue_batched_commands kernel failed");

    hipStreamSynchronize(stream1);
    cudaCheckErrors("hipStreamSynchronize failed");
    auto t1 = std::chrono::high_resolution_clock::now();

    hipDeviceSynchronize();
    cudaCheckErrors("hipDeviceSynchronize failed");

    printf("Before cpu_thraed join\n");
    for (auto& t : cpu_threads) {
      t.join();
    }
    printf("After cpu_thraed join\n");

    unsigned int tot_ops = 0;
#ifdef MEASURE_PER_OP_LATENCY
    double total_us = 0;
    unsigned long long tot_cycles = 0;
    printf("\nPer-block avg latency:\n");
    for (int b = 0; b < kNumThBlocks; ++b) {
      double us = (double)rbs[b].cycle_accum * 1000.0 / prop.clockRate /
                  rbs[b].op_count;
      printf("  Block %d : %.3f µs over %lu ops\n", b, us, rbs[b].op_count);
      total_us += us;
      tot_cycles += rbs[b].cycle_accum;
      tot_ops += rbs[b].op_count;
    }
#else
    tot_ops = kNumThBlocks * kIterations;
#endif
    double wall_ms = std::chrono::duration<double, std::milli>(t1 - t0).count();
    double throughput =
        (double)(kNumThBlocks * kIterations) / (wall_ms * 1000.0);

#ifdef MEASURE_PER_OP_LATENCY
    printf("\nOverall avg GPU-measured latency  : %.3f µs\n",
           (double)tot_cycles * 1000.0 / prop.clockRate / tot_ops);
    printf("Total cycles                       : %llu\n", tot_cycles);
#endif
    printf("Total ops                          : %u\n", tot_ops);
    printf("End-to-end Wall-clock time        : %.3f ms\n", wall_ms);
    printf("Throughput                        : %.2f Mops/s\n", throughput);

    hipHostFree(rbs);
    cudaCheckErrors("hipHostFree failed");
    hipStreamDestroy(stream1);
    cudaCheckErrors("hipStreamDestroy failed");
  } else {
    // Infinite loop
    while (true) {
      std::this_thread::sleep_for(std::chrono::seconds(1));
      printf("Rank %d is waiting...\n", rank);
    }
  }
}