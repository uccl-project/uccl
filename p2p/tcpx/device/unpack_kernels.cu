#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2024, UCCL Project. All rights reserved.
 * Adapted from NCCL unpack kernel logic
 *
 * See LICENSE.txt for license information
 ************************************************************************/
#include "../include/unpack_descriptor.h"
#include "unpack_launch.h"
#include <algorithm>
#include <cstdint>
#include <hip/hip_runtime.h>

// Forward declaration for device visibility barrier used by probe kernel
namespace tcpx {
namespace device {
__device__ __forceinline__ void devmem_visibility_barrier(void const* flag_ptr);
}
}  // namespace tcpx

namespace tcpx {
// Define a minimal staging kernel for debugging: read 1 byte and write to
// dst[0]
extern "C" __global__ void tcpxUnpackKernelProbeByte(
    tcpx::rx::UnpackDescriptorBlock const* desc_block) {
  // Only thread 0 does the visibility load, but all threads must sync
  if (threadIdx.x == 0) {
    device::devmem_visibility_barrier(desc_block->ready_flag);
  }
  __syncthreads();  // All threads sync here

  if (blockIdx.x == 0 && threadIdx.x == 0 && desc_block->count > 0) {
    tcpx::rx::UnpackDescriptor const& d = desc_block->descriptors[0];
    char const* src =
        static_cast<char const*>(desc_block->bounce_buffer) + d.src_off;
    char* dst = static_cast<char*>(desc_block->dst_buffer) + d.dst_off;
    char volatile v = *src;
    *dst = v;
  }
}

namespace device {

// CUDA memory access primitives (adapted from NCCL)
template <int BYTES>
struct BytePack;

template <>
struct BytePack<1> {
  uint8_t u8;
};
template <>
struct BytePack<2> {
  uint16_t u16;
};
template <>
struct BytePack<4> {
  uint32_t u32;
};
template <>
struct BytePack<8> {
  uint64_t u64;
};
template <>
struct BytePack<16> {
  uint64_t u64[2];
};

// Vectorized load/store functions
template <int BYTES>
__device__ __forceinline__ BytePack<BYTES> ld_volatile_global(uintptr_t addr);

template <>
__device__ __forceinline__ BytePack<1> ld_volatile_global<1>(uintptr_t addr) {
  BytePack<1> val;
  val.u8 = *reinterpret_cast<uint8_t volatile*>(addr);
  return val;
}

template <>
__device__ __forceinline__ BytePack<2> ld_volatile_global<2>(uintptr_t addr) {
  BytePack<2> val;
  val.u16 = *reinterpret_cast<uint16_t volatile*>(addr);
  return val;
}

template <>
__device__ __forceinline__ BytePack<4> ld_volatile_global<4>(uintptr_t addr) {
  BytePack<4> val;
  val.u32 = *reinterpret_cast<uint32_t volatile*>(addr);
  return val;
}

template <>
__device__ __forceinline__ BytePack<8> ld_volatile_global<8>(uintptr_t addr) {
  BytePack<8> val;
  val.u64 = *reinterpret_cast<uint64_t volatile*>(addr);
  return val;
}

template <>
__device__ __forceinline__ BytePack<16> ld_volatile_global<16>(uintptr_t addr) {
  BytePack<16> val;
  const volatile uint64_t* ptr =
      reinterpret_cast<const volatile uint64_t*>(addr);
  val.u64[0] = ptr[0];
  val.u64[1] = ptr[1];
  return val;
}

template <int BYTES>
__device__ __forceinline__ void st_global(uintptr_t addr, BytePack<BYTES> val);

template <>
__device__ __forceinline__ void st_global<1>(uintptr_t addr, BytePack<1> val) {
  *reinterpret_cast<uint8_t*>(addr) = val.u8;
}

template <>
__device__ __forceinline__ void st_global<2>(uintptr_t addr, BytePack<2> val) {
  *reinterpret_cast<uint16_t*>(addr) = val.u16;
}

template <>
__device__ __forceinline__ void st_global<4>(uintptr_t addr, BytePack<4> val) {
  *reinterpret_cast<uint32_t*>(addr) = val.u32;
}

template <>
__device__ __forceinline__ void st_global<8>(uintptr_t addr, BytePack<8> val) {
  *reinterpret_cast<uint64_t*>(addr) = val.u64;
}

template <>
__device__ __forceinline__ void st_global<16>(uintptr_t addr,
                                              BytePack<16> val) {
  uint64_t* ptr = reinterpret_cast<uint64_t*>(addr);
  ptr[0] = val.u64[0];
  ptr[1] = val.u64[1];
}

// Constants
#define DATA_LOAD_SIZE 16
#define WARP_SIZE 32
constexpr int kWarpShmPageCnt = 4;

// Device-side visibility barrier similar to NCCL load64gpu on cnt
// NOTE: This function does NOT include __syncthreads() - caller must sync if
// needed
__device__ __forceinline__ void devmem_visibility_barrier(
    void const* flag_ptr) {
  if (!flag_ptr) return;
#if __CUDA_ARCH__ >= 700
  unsigned long long v;
  asm volatile("ld.relaxed.gpu.u64 {%0}, [%1];"
               : "=l"(v)
               : "l"(flag_ptr)
               : "memory");
#else
  unsigned long long volatile* p = (unsigned long long volatile*)flag_ptr;
  (void)*p;
#endif
}

// Bulk copy template (adapted from NCCL)
// IMPORTANT: Loop condition must match NCCL exactly: data_s + DATA_LOAD_SIZE -
// 1 < len
template <int BYTES>
__device__ void bulkCopy(int lane, uint32_t len, char* src, char* dst) {
  int const elements_per_thread = DATA_LOAD_SIZE / BYTES;
  BytePack<BYTES> reg[elements_per_thread];

  // Match NCCL's loop condition exactly
  for (uint32_t offset = lane * DATA_LOAD_SIZE;
       offset + DATA_LOAD_SIZE - 1 <
       len;  // Fixed: was "offset + DATA_LOAD_SIZE <= len"
       offset += WARP_SIZE * DATA_LOAD_SIZE) {
// Load data
#pragma unroll
    for (int i = 0; i < elements_per_thread; ++i) {
      reg[i] = ld_volatile_global<BYTES>(
          reinterpret_cast<uintptr_t>(src + offset) + i * BYTES);
    }

// Store data
#pragma unroll
    for (int i = 0; i < elements_per_thread; ++i) {
      st_global<BYTES>(reinterpret_cast<uintptr_t>(dst + offset) + i * BYTES,
                       reg[i]);
    }
  }
}

// Single descriptor unpack kernel
// Matches NCCL logic from unpack.h:251-275
__device__ void unpackSingleDescriptor(int lane,
                                       tcpx::rx::UnpackDescriptor const& desc,
                                       char* bounce_buffer, char* dst_buffer) {
  char* src = bounce_buffer + desc.src_off;
  char* dst = dst_buffer + desc.dst_off;
  uint32_t len = desc.len;

  // Fast path for data >= 16 bytes (matches NCCL line 251)
  if (len >= DATA_LOAD_SIZE) {
    // Determine optimal alignment for vectorized access (matches NCCL line
    // 255-256)
    uint8_t align_off = (desc.src_off | desc.dst_off) % DATA_LOAD_SIZE;
    align_off = align_off & (-align_off);  // Keep lowest bit

    // Select bulk copy size based on alignment (matches NCCL line 257-267)
    if (align_off == 0) {
      bulkCopy<16>(lane, len, src, dst);
    } else if (align_off & 0x8) {
      bulkCopy<8>(lane, len, src, dst);
    } else if (align_off & 0x4) {
      bulkCopy<4>(lane, len, src, dst);
    } else if (align_off & 0x2) {
      bulkCopy<2>(lane, len, src, dst);
    } else {
      bulkCopy<1>(lane, len, src, dst);
    }
  }

  // Handle remaining bytes (< DATA_LOAD_SIZE) (matches NCCL line 271-275)
  // This handles both:
  // 1. Tail bytes after bulk copy (e.g., 23 bytes = 16 bulk + 7 tail)
  // 2. Entire payload if len < 16 (e.g., 7 bytes = 0 bulk + 7 tail)
  uint32_t remaining_start = (len / DATA_LOAD_SIZE) * DATA_LOAD_SIZE;
  if (lane < len % DATA_LOAD_SIZE) {
    char volatile* src_ptr = src + remaining_start + lane;
    char volatile* dst_ptr = dst + remaining_start + lane;
    *dst_ptr = *src_ptr;
  }
}

// Main unpack kernel
extern "C" __global__ void tcpxUnpackKernel(
    tcpx::rx::UnpackDescriptorBlock const* desc_block) {
  int tid = threadIdx.x;

  if (tid == 0) {
    devmem_visibility_barrier(desc_block->ready_flag);
  }
  __syncthreads();

  char* bounce_buffer = static_cast<char*>(desc_block->bounce_buffer);
  char* dst_buffer = static_cast<char*>(desc_block->dst_buffer);

  int const lane = tid & (WARP_SIZE - 1);
  int const warp_id = tid / WARP_SIZE;
  int const warps_per_block = blockDim.x / WARP_SIZE;
  int const total_warps = gridDim.x * warps_per_block;

  extern __shared__ unsigned char smem[];
  auto* warp_cache = reinterpret_cast<tcpx::rx::UnpackDescriptor*>(smem) +
                     warp_id * kWarpShmPageCnt;

  int const warp_global = blockIdx.x * warps_per_block + warp_id;
  for (int base = warp_global * kWarpShmPageCnt; base < desc_block->count;
       base += total_warps * kWarpShmPageCnt) {
    int batch = desc_block->count - base;
    if (batch > kWarpShmPageCnt) batch = kWarpShmPageCnt;
    if (batch <= 0) continue;

    if (lane < batch) {
      warp_cache[lane] = desc_block->descriptors[base + lane];
    }
    __syncwarp();

    for (int i = 0; i < batch; ++i) {
      tcpx::rx::UnpackDescriptor const& desc = warp_cache[i];
      unpackSingleDescriptor(lane, desc, bounce_buffer, dst_buffer);
    }
    __syncwarp();
  }
}

// Optimized kernel for small descriptors (single warp per descriptor)
// Note: Unlike NCCL which has warps process multiple descriptors in a loop,
// we assign one warp per descriptor. No inter-warp sync needed since each
// warp works on independent memory regions (different dst_off ranges).
extern "C" __global__ void tcpxUnpackKernelSmall(
    tcpx::rx::UnpackDescriptorBlock const* desc_block) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int warp_id = tid / WARP_SIZE;
  int lane_id = tid % WARP_SIZE;

  // Issue a device-side visibility barrier before reading bounce buffer
  // Only thread 0 in each block does the visibility load, but all threads must
  // sync
  if (threadIdx.x == 0) {
    devmem_visibility_barrier(desc_block->ready_flag);
  }
  __syncthreads();  // All threads in block sync here

  char* bounce_buffer = static_cast<char*>(desc_block->bounce_buffer);
  char* dst_buffer = static_cast<char*>(desc_block->dst_buffer);

  // Each warp processes one descriptor
  // No sync needed after processing since descriptors have non-overlapping
  // dst_off ranges
  if (warp_id < desc_block->count) {
    tcpx::rx::UnpackDescriptor const& desc = desc_block->descriptors[warp_id];
    unpackSingleDescriptor(lane_id, desc, bounce_buffer, dst_buffer);
  }
}

// Kernel launch parameters calculation (use header definition)

// Calculate optimal launch parameters
extern "C" __host__ tcpx::device::KernelLaunchParams calculateLaunchParams(
    tcpx::rx::UnpackDescriptorBlock const& desc_block) {
  tcpx::device::KernelLaunchParams params;

  if (desc_block.count == 0) {
    params.grid_size = dim3(0);
    params.block_size = dim3(0);
    return params;
  }

  // Determine if we should use small kernel (warp-per-descriptor)
  bool use_small_kernel = true;
  for (uint32_t i = 0; i < desc_block.count; ++i) {
    if (desc_block.descriptors[i].len > 1024) {
      use_small_kernel = false;
      break;
    }
  }

  if (use_small_kernel) {
    // Warp per descriptor
    int warps_needed = desc_block.count;
    int threads_per_block = 256;  // 8 warps per block
    int blocks_needed =
        (warps_needed * WARP_SIZE + threads_per_block - 1) / threads_per_block;

    params.grid_size = dim3(blocks_needed);
    params.block_size = dim3(threads_per_block);
    params.shared_mem_size = 0;
  } else {
    // Block per descriptor
    params.grid_size = dim3(std::max<uint32_t>(1, desc_block.count));
    params.block_size = dim3(256);  // 8 warps per block
    int warps_per_block = params.block_size.x / WARP_SIZE;
    params.shared_mem_size =
        warps_per_block * kWarpShmPageCnt * sizeof(tcpx::rx::UnpackDescriptor);
  }
  params.use_small_kernel = use_small_kernel;

  return params;
}

}  // namespace device
}  // namespace tcpx
