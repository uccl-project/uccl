#include <hip/hip_runtime.h>

#include <chrono>
#include <iostream>
#include <vector>

#define NUM_4KB_BLOCKS 32  // Adjust this for different block counts
#define BLOCK_SIZE 4096    // 4KB
#define NUM_STREAMS 4      // Number of CUDA streams for async copy

void checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << " - " << hipGetErrorString(err)
                  << std::endl;
        exit(EXIT_FAILURE);
    }
}

__global__ void initDataKernel(char **scattered_data, int num_blocks) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < num_blocks) {
        for (int i = 0; i < BLOCK_SIZE; i++) {
            scattered_data[idx][i] = idx % 256;  // Fill with some pattern
        }
    }
}

__global__ void copyKernel(char **scattered_data, char *continuous_data,
                           int num_blocks) {
    // Each thread copies exactly one byte from scattered_data to
    // continuous_data.
    int globalId = blockIdx.x * blockDim.x + threadIdx.x;
    int totalSize = num_blocks * BLOCK_SIZE;  // total bytes

    if (globalId < totalSize) {
        // Determine which 4KB block this byte belongs to
        int blockIndex = globalId / BLOCK_SIZE;  // which block
        int offset = globalId % BLOCK_SIZE;      // offset in block

        continuous_data[globalId] = scattered_data[blockIndex][offset];
    }
}

void benchmarkMemcpy() {
    // Allocate scattered 4KB memory blocks on GPU
    char **d_scattered_ptrs;
    checkCudaError(
        hipMalloc(&d_scattered_ptrs, NUM_4KB_BLOCKS * sizeof(char *)),
        "Allocating scattered pointer array");

    std::vector<char *> d_scattered(NUM_4KB_BLOCKS);
    for (int i = 0; i < NUM_4KB_BLOCKS; i++) {
        checkCudaError(hipMalloc(&d_scattered[i], BLOCK_SIZE),
                       "Allocating scattered data block");
    }
    checkCudaError(
        hipMemcpy(d_scattered_ptrs, d_scattered.data(),
                   NUM_4KB_BLOCKS * sizeof(char *), hipMemcpyHostToDevice),
        "Copying pointer list to device");

    // Allocate continuous GPU buffer
    char *d_continuous;
    checkCudaError(hipMalloc(&d_continuous, NUM_4KB_BLOCKS * BLOCK_SIZE),
                   "Allocating continuous buffer");

    // Initialize scattered data
    initDataKernel<<<(NUM_4KB_BLOCKS + 255) / 256, 256>>>(d_scattered_ptrs,
                                                          NUM_4KB_BLOCKS);
    hipDeviceSynchronize();

    // Benchmark cudaMemcpy
    auto start_time = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < NUM_4KB_BLOCKS; i++) {
        checkCudaError(hipMemcpy(d_continuous + i * BLOCK_SIZE, d_scattered[i],
                                  BLOCK_SIZE, hipMemcpyDeviceToDevice),
                       "Memcpy scattered to continuous");
    }

    auto stop_time = std::chrono::high_resolution_clock::now();
    auto elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>(
        stop_time - start_time);

    std::cout << "hipMemcpy performance: "
              << (NUM_4KB_BLOCKS * BLOCK_SIZE) / (elapsed_time.count() * 1e3)
              << " GB/s" << std::endl;

    // Benchmark cudaMemcpyAsync with multiple streams
    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        checkCudaError(hipStreamCreate(&streams[i]), "Creating stream");
    }

    start_time = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < NUM_4KB_BLOCKS; i++) {
        int stream_id =
            i % NUM_STREAMS;  // Assign each copy operation to a stream
        checkCudaError(
            hipMemcpyAsync(d_continuous + i * BLOCK_SIZE, d_scattered[i],
                            BLOCK_SIZE, hipMemcpyDeviceToDevice,
                            streams[stream_id]),
            "MemcpyAsync scattered to continuous");
    }

    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamSynchronize(streams[i]);
    }

    stop_time = std::chrono::high_resolution_clock::now();
    elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>(
        stop_time - start_time);

    std::cout << "hipMemcpyAsync performance: "
              << (NUM_4KB_BLOCKS * BLOCK_SIZE) / (elapsed_time.count() * 1e3)
              << " GB/s" << std::endl;

    // --------------------------------------------------
    // Benchmark the GPU kernel copy
    // --------------------------------------------------
    start_time = std::chrono::high_resolution_clock::now();

    // 1D grid to cover all bytes: (NUM_4KB_BLOCKS * BLOCK_SIZE) total
    int totalBytes = NUM_4KB_BLOCKS * BLOCK_SIZE;
    int threadsPerBlock = 256;
    int gridSize = (totalBytes + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel to copy each byte in parallel
    copyKernel<<<gridSize, threadsPerBlock>>>(d_scattered_ptrs, d_continuous,
                                              NUM_4KB_BLOCKS);

    // Wait for the kernel to finish so timing is accurate
    checkCudaError(hipDeviceSynchronize(), "Synchronize after copyKernel");

    stop_time = std::chrono::high_resolution_clock::now();
    auto elapsed_time_us =
        std::chrono::duration_cast<std::chrono::microseconds>(stop_time -
                                                              start_time);

    double elapsed_time_s =
        static_cast<double>(elapsed_time_us.count()) * 1e-6;  // seconds
    double total_bytes =
        static_cast<double>(NUM_4KB_BLOCKS) * BLOCK_SIZE;  // total bytes
    double bandwidth_gb_s =
        (total_bytes / (1024.0 * 1024.0 * 1024.0)) / elapsed_time_s;  // in GB/s

    std::cout << "Kernel copy performance (" << NUM_4KB_BLOCKS
              << " 4KB pages): " << elapsed_time_us.count() << " us, or "
              << bandwidth_gb_s << " GB/s" << std::endl;

    // Cleanup
    for (int i = 0; i < NUM_4KB_BLOCKS; i++) {
        hipFree(d_scattered[i]);
    }
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamDestroy(streams[i]);
    }
    hipFree(d_scattered_ptrs);
    hipFree(d_continuous);
}

int main() {
    hipSetDevice(0);
    benchmarkMemcpy();
    return 0;
}
