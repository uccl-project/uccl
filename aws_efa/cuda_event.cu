#include <hip/hip_runtime.h>

#include <iostream>

// An empty kernel that does nothing.
__global__ void emptyKernel() {}

int main() {
    const int iterations = 1000;
    float totalTimeMs = 0.0f;
    hipError_t err;

    // Create CUDA events for timing.
    hipEvent_t start, stop;
    err = hipEventCreate(&start);
    if (err != hipSuccess) {
        std::cerr << "Error creating start event: " << hipGetErrorString(err)
                  << std::endl;
        return -1;
    }
    err = hipEventCreate(&stop);
    if (err != hipSuccess) {
        std::cerr << "Error creating stop event: " << hipGetErrorString(err)
                  << std::endl;
        return -1;
    }

    // Warm up: Launch the kernel once and synchronize.
    emptyKernel<<<1, 1>>>();
    hipDeviceSynchronize();

    // Run the measurement loop.
    for (int i = 0; i < iterations; i++) {
        // Record the start event.
        hipEventRecord(start, 0);
        // Launch the empty kernel.
        emptyKernel<<<1, 1>>>();
        // Record the stop event.
        hipEventRecord(stop, 0);
        // Wait for the stop event to complete.
        hipEventSynchronize(stop);

        // Measure elapsed time in milliseconds.
        float elapsedMs = 0.0f;
        hipEventElapsedTime(&elapsedMs, start, stop);
        totalTimeMs += elapsedMs;
    }

    // Calculate average time in milliseconds.
    double averageTimeMs = totalTimeMs / iterations;
    // Convert average time to microseconds.
    double averageTimeUs = averageTimeMs * 1000.0;
    std::cout << "Average empty kernel launch time over " << iterations
              << " iterations: " << averageTimeUs << " microseconds"
              << std::endl;

    // Clean up.
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
