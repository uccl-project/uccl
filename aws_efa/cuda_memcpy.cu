#include <hip/hip_runtime.h>

#include <chrono>
#include <iostream>

int main() {
    const size_t size = 4096;  // 4KB copy
    const int iterations = 1000;

    // Allocate memory on the device.
    void* d_buf;
    hipError_t err = hipMalloc(&d_buf, size);
    if (err != hipSuccess) {
        std::cerr << "hipMalloc error: " << hipGetErrorString(err)
                  << std::endl;
        return 1;
    }

    // Allocate pinned host memory.
    void* h_buf;
    err = hipHostMalloc(&h_buf, size, hipHostMallocDefault);
    if (err != hipSuccess) {
        std::cerr << "hipHostMalloc error: " << hipGetErrorString(err)
                  << std::endl;
        hipFree(d_buf);
        return 1;
    }

    // Warm-up: perform one copy to eliminate any one-time overhead.
    hipMemcpy(h_buf, d_buf, size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Measure the cudaMemcpy time over many iterations.
    auto start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < iterations; i++) {
        // Copy from device to host.
        hipMemcpy(h_buf, d_buf, size, hipMemcpyDeviceToHost);
        // Synchronize to ensure the copy is complete before measuring time.
        hipDeviceSynchronize();
    }
    auto end = std::chrono::high_resolution_clock::now();

    // Calculate elapsed time.
    auto total_ns =
        std::chrono::duration_cast<std::chrono::nanoseconds>(end - start)
            .count();
    double average_time_ns = static_cast<double>(total_ns) / iterations;
    std::cout << "Average hipMemcpy time: " << average_time_ns << " ns"
              << std::endl;

    // Clean up.
    hipFree(d_buf);
    hipHostFree(h_buf);

    return 0;
}
