#include <iostream>
#include <thread>
#include <assert.h>
#include <hip/hip_runtime.h>


#define CHECK_CUDA(call)                                                     \
  do {                                                                       \
    hipError_t result = call;                                                  \
    if (result != hipSuccess) {                                            \
      char const* errorString;                                               \
      hipDrvGetErrorString(result, &errorString);                                \
      fprintf(stderr, "CUDA Error at line %d: %s\n", __LINE__, errorString); \
      exit(1);                                                               \
    }                                                                        \
  } while (0)

#define N 1024  // Number of elements

__device__ uint __smid(void) {
  uint ret;
  asm("mov.u32 %0, %smid;" : "=r"(ret));
  return ret;
}

// Dummy kernel to simulate computation
__global__ void kernel1(int* data) {
  // if (threadIdx.x == 0) printf("Kernel1 SM %d\n", __smid());
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  for (uint64_t i = 0; i < 10000000u; i++) {  // Simulate workload
    data[idx] *= 2 * i;
  }
}

// Another dummy kernel
__global__ void kernel2(int* data) {
  // if (threadIdx.x == 0) printf("Kernel2 SM %d\n", __smid());
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = 0; i < 1000000; i++) {  // Simulate workload
    data[idx] *= 2 * i;
  }
}

unsigned int const kNbGroups = 2;        // Desired number of groups
CUdevResource resources[kNbGroups + 1];  // Array to store split resources
hipCtx_t context[kNbGroups + 1];

void PartitionGreenCtx() {
  hipDevice_t device;

  CHECK_CUDA(hipInit(0));
  CHECK_CUDA(hipDeviceGet(&device, 0));

  // now, opt into green context
  CUdevResource sm_resource;
  CHECK_CUDA(
      cuDeviceGetDevResource(device, &sm_resource, CU_DEV_RESOURCE_TYPE_SM));
  printf("SM Resource: %d\n", sm_resource.sm.smCount);

  // Split the SM resource
  unsigned int minCount = 8;  // Minimum SMs per group
  uint32_t nbGroups = kNbGroups;
  CHECK_CUDA(cuDevSmResourceSplitByCount(resources, &nbGroups, &sm_resource,
                                         &resources[kNbGroups], 0, minCount));
  printf("Number of groups created: %d\n", nbGroups + 1);
  assert(nbGroups == kNbGroups);

  for (int i = 0; i < nbGroups + 1; i++) {
    printf("Group %d: %d SMs\n", i, resources[i].sm.smCount);

    // generate descriptor for the first group
    CUdevResourceDesc desc;
    CHECK_CUDA(cuDevResourceGenerateDesc(&desc, &resources[i], 1));

    CUgreenCtx green_ctx;
    CHECK_CUDA(cuGreenCtxCreate(&green_ctx, desc, device,
                                CU_GREEN_CTX_DEFAULT_STREAM));

    CUdevResource green_sm_resource;
    CHECK_CUDA(cuGreenCtxGetDevResource(green_ctx, &green_sm_resource,
                                        CU_DEV_RESOURCE_TYPE_SM));
    printf("Green SM Resource: %d\n", green_sm_resource.sm.smCount);

    CHECK_CUDA(cuCtxFromGreenCtx(&context[i], green_ctx));
  }
}

// nvcc -o cuda_concurrent cuda_concurrent.cu -lcuda -lcudart

int main() {
  PartitionGreenCtx();

  hipStream_t stream1, stream2;
  int *d_data1, *d_data2;

  hipCtxSetCurrent(context[0]);
  hipMalloc(&d_data1, N * sizeof(int));
  hipStreamCreate(&stream1);

  hipCtxSetCurrent(context[1]);
  hipMalloc(&d_data2, N * sizeof(int));
  hipStreamCreate(&stream2);

  // Launching convention #1
  hipCtxSetCurrent(context[0]);
  for (int i = 0; i < 10; i++) {
    kernel1<<<4, 256, 0, stream1>>>(d_data1);
  }
  hipCtxSetCurrent(context[1]);
  for (int i = 0; i < 10; i++) {
    kernel2<<<1, 256, 0, stream2>>>(d_data2);
  }
  hipStreamSynchronize(stream1);
  hipStreamSynchronize(stream2);

  // Launching convention #2
  for (int i = 0; i < 10; i++) {
    hipCtxSetCurrent(context[0]);
    kernel1<<<4, 256, 0, stream1>>>(d_data1);
    hipCtxSetCurrent(context[1]);
    kernel2<<<1, 256, 0, stream2>>>(d_data2);
  }
  hipStreamSynchronize(stream1);
  hipStreamSynchronize(stream2);

  // Cleanup
  hipFree(d_data1);
  hipFree(d_data2);
  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);

  std::cout << "✅ Both kernels executed concurrently!" << std::endl;
  return 0;
}
