#include <chrono>
#include <iostream>
#include <hip/hip_runtime.h>

// Average async cudaMemcpy submission overhead: 2407.53 ns

int main() {
  const size_t size = 4096;  // 4 KB buffer size.
  int const iterations = 10000;

  // Allocate device memory.
  void* d_buf = nullptr;
  hipError_t err = hipMalloc(&d_buf, size);
  if (err != hipSuccess) {
    std::cerr << "hipMalloc error: " << hipGetErrorString(err) << std::endl;
    return 1;
  }

  // Allocate pinned (page-locked) host memory.
  void* h_buf = nullptr;
  err = hipHostMalloc(&h_buf, size, hipHostMallocDefault);
  if (err != hipSuccess) {
    std::cerr << "hipHostMalloc error: " << hipGetErrorString(err)
              << std::endl;
    hipFree(d_buf);
    return 1;
  }

  // Create a stream for asynchronous copies.
  hipStream_t stream;
  err = hipStreamCreate(&stream);
  if (err != hipSuccess) {
    std::cerr << "hipStreamCreate error: " << hipGetErrorString(err)
              << std::endl;
    hipFree(d_buf);
    hipHostFree(h_buf);
    return 1;
  }

  // Warm up the GPU/driver to avoid one-time initialization overhead.
  hipMemcpyAsync(h_buf, d_buf, size, hipMemcpyDeviceToHost, stream);
  hipStreamSynchronize(stream);

  // Use std::chrono to measure the CPU overhead of submitting async copies.
  auto start = std::chrono::high_resolution_clock::now();
  for (int i = 0; i < iterations; i++) {
    // Submit an asynchronous copy from device to host.
    hipMemcpyAsync(h_buf, d_buf, size, hipMemcpyDeviceToHost, stream);
    // Note: We do not call cudaStreamSynchronize() here so that we only
    // measure the submission cost.
  }
  auto end = std::chrono::high_resolution_clock::now();

  // Calculate the total elapsed time in nanoseconds.
  auto total_ns =
      std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  double avg_submission_time_ns = static_cast<double>(total_ns) / iterations;
  std::cout << "Average async hipMemcpy submission overhead: "
            << avg_submission_time_ns << " ns" << std::endl;

  // Synchronize the stream to complete all pending operations (not included
  // in the timing).
  hipStreamSynchronize(stream);

  // Cleanup.
  hipStreamDestroy(stream);
  hipFree(d_buf);
  hipHostFree(h_buf);

  // Measure cudaMemcpyPeerAsync overhead between two GPUs
  int device_count;
  err = hipGetDeviceCount(&device_count);
  if (err != hipSuccess || device_count < 2) {
    std::cout << "Need at least 2 GPUs to measure peer copy overhead. Found "
              << device_count << " GPUs." << std::endl;
    return 0;
  }

  // Allocate memory on two different GPUs
  void *buf0 = nullptr, *buf1 = nullptr;
  hipSetDevice(0);
  err = hipMalloc(&buf0, size);
  if (err != hipSuccess) {
    std::cerr << "hipMalloc error on GPU 0: " << hipGetErrorString(err)
              << std::endl;
    return 1;
  }

  hipSetDevice(1);
  err = hipMalloc(&buf1, size);
  if (err != hipSuccess) {
    std::cerr << "hipMalloc error on GPU 1: " << hipGetErrorString(err)
              << std::endl;
    hipFree(buf0);
    return 1;
  }

  // Enable peer access between the GPUs
  hipSetDevice(0);
  err = hipDeviceEnablePeerAccess(1, 0);
  if (err != hipSuccess && err != hipErrorPeerAccessAlreadyEnabled) {
    std::cerr << "Failed to enable peer access 0->1: "
              << hipGetErrorString(err) << std::endl;
    hipFree(buf0);
    hipFree(buf1);
    return 1;
  }

  // Create a new stream for peer copies
  hipStream_t peer_stream;
  err = hipStreamCreate(&peer_stream);
  if (err != hipSuccess) {
    std::cerr << "hipStreamCreate error: " << hipGetErrorString(err)
              << std::endl;
    hipFree(buf0);
    hipFree(buf1);
    return 1;
  }

  // Warm up
  hipMemcpyPeerAsync(buf1, 1, buf0, 0, size, peer_stream);
  hipStreamSynchronize(peer_stream);

  // Measure peer copy submission overhead
  start = std::chrono::high_resolution_clock::now();
  for (int i = 0; i < iterations; i++) {
    hipMemcpyPeerAsync(buf1, 1, buf0, 0, size, peer_stream);
  }
  end = std::chrono::high_resolution_clock::now();

  total_ns =
      std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  double avg_peer_submission_time_ns =
      static_cast<double>(total_ns) / iterations;
  std::cout << "Average async hipMemcpyPeer submission overhead: "
            << avg_peer_submission_time_ns << " ns" << std::endl;

  // Cleanup
  hipStreamSynchronize(peer_stream);
  hipStreamDestroy(peer_stream);
  hipDeviceDisablePeerAccess(1);
  hipFree(buf0);
  hipFree(buf1);

  return 0;
}
