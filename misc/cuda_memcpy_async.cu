#include <chrono>
#include <iostream>
#include <hip/hip_runtime.h>

// Average async cudaMemcpy submission overhead: 2407.53 ns

int main() {
  const size_t size = 4096;  // 4 KB buffer size.
  int const iterations = 10000;

  // Allocate device memory.
  void* d_buf = nullptr;
  hipError_t err = hipMalloc(&d_buf, size);
  if (err != hipSuccess) {
    std::cerr << "hipMalloc error: " << hipGetErrorString(err) << std::endl;
    return 1;
  }

  // Allocate pinned (page-locked) host memory.
  void* h_buf = nullptr;
  err = hipHostMalloc(&h_buf, size, hipHostMallocDefault);
  if (err != hipSuccess) {
    std::cerr << "hipHostMalloc error: " << hipGetErrorString(err)
              << std::endl;
    hipFree(d_buf);
    return 1;
  }

  // Create a stream for asynchronous copies.
  hipStream_t stream;
  err = hipStreamCreate(&stream);
  if (err != hipSuccess) {
    std::cerr << "hipStreamCreate error: " << hipGetErrorString(err)
              << std::endl;
    hipFree(d_buf);
    hipHostFree(h_buf);
    return 1;
  }

  // Warm up the GPU/driver to avoid one-time initialization overhead.
  hipMemcpyAsync(h_buf, d_buf, size, hipMemcpyDeviceToHost, stream);
  hipStreamSynchronize(stream);

  // Use std::chrono to measure the CPU overhead of submitting async copies.
  auto start = std::chrono::high_resolution_clock::now();
  for (int i = 0; i < iterations; i++) {
    // Submit an asynchronous copy from device to host.
    hipMemcpyAsync(h_buf, d_buf, size, hipMemcpyDeviceToHost, stream);
    // Note: We do not call cudaStreamSynchronize() here so that we only
    // measure the submission cost.
  }
  auto end = std::chrono::high_resolution_clock::now();

  // Calculate the total elapsed time in nanoseconds.
  auto total_ns =
      std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  double avg_submission_time_ns = static_cast<double>(total_ns) / iterations;
  std::cout << "Average async hipMemcpy submission overhead: "
            << avg_submission_time_ns << " ns" << std::endl;

  // Synchronize the stream to complete all pending operations (not included
  // in the timing).
  hipStreamSynchronize(stream);

  // Cleanup.
  hipStreamDestroy(stream);
  hipFree(d_buf);
  hipHostFree(h_buf);

  return 0;
}
